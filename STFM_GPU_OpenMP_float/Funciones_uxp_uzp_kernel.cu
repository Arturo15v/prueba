#include "hip/hip_runtime.h"

__global__ void Para_uxp_uzp(float *p_dev,float *Ox_dev,float *Oz_dev,float *ux_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,float *vel_dev,
						  int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						  float tx2,float dh,
						  int N,int M,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2/*, int lim2*/){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	float dl,da,pi,cf,aa,ak;
	int h,k;

	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
			int ind_p=k2*Nzz*Nxx+idy*Nxx;
			int ind_Ox=Nzz*Na+ idy*Na;
			int ind_ux=idx + idy*Na;
			int ind_ux2=Nzz*Na + idx + idy*Na;
			int ind_p2=k1*Nzz*Nxx + idx + idy*Nxx;
			int ind_p3=k1*Nzz*Nxx+idy*Nxx;

			dl=g_dev[ms]*p_dev[ind_p + idx];       
			pi=g_dev[ms]*p_dev[ind_p + (Nxx-idx)];
			da=0.0;                   cf=0.0;

			for(h=1;h<=mn_dev[idx];h++){
				dl=dl+g_dev[h]*(p_dev[ind_p + (idx-h)]+p_dev[ind_p + (idx+h)]);
				da=da-d_dev[h]*Ox_dev[(idx-h) + idy*Na]*Tn_dev[(idx-h)];
				pi=pi+g_dev[h]*(p_dev[ind_p + ((Nxx-idx)-h)]+p_dev[ind_p + ((Nxx-idx)+h)]);
				cf=cf+d_dev[h]*Ox_dev[ind_Ox + (idx-h)]*Tn_dev[(idx-h)];	
			}
			for(k=h;k<=no;k++){
				dl=dl+g_dev[k]*p_dev[ind_p + (idx+k)];
				pi=pi+g_dev[k]*p_dev[ind_p + ((Nxx-idx)-k)];
			}
			for(h=1;h<=ma_dev[idx];h++){
				da=da+d_dev[h]*Ox_dev[(idx+h) + idy*Na]*Tn_dev[(idx+h)];
				cf=cf-d_dev[h]*Ox_dev[ind_Ox + (idx+h)]*Tn_dev[(idx+h)];
			}
			aa=dl - da*dh - Tk_dev[idx]*ux_dev[ind_ux];
			ux_dev[ind_ux]=tx2*aa+ux_dev[ind_ux];
			p_dev[ind_p2]=vel_dev[idy*Nxx + idx]*aa+ 2.0f*p_dev[ind_p + idx] - p_dev[ind_p2];
			aa=pi - cf*dh - Tk_dev[idx]*ux_dev[ind_ux2];
			ux_dev[ind_ux2]=tx2*aa+ux_dev[ind_ux2];
			p_dev[ind_p3 + (Nxx-idx)]=vel_dev[idy*Nxx + (Nxx-idx)]*aa + 
				                          2.0f*p_dev[ind_p + (Nxx-idx)] - p_dev[ind_p3 + (Nxx-idx)];

	}
	if (idx>0 && idx<Na && idy>0 && idy<Na){
		int ind_p1=k1*Nzz*Nxx + idy;
		int ind_p2=k2*Nzz*Nxx + idy;
		int ind_Oz=idy;

		pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
		cf=0.0;
		for(h=1;h<=mn_dev[idx];h++){
			pi=pi+g_dev[h]*(p_dev[ind_p2 + ((Nzz-idx)-h)*Nxx]+p_dev[ind_p2 + ((Nzz-idx)+h)*Nxx]);
			cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];

		}
		for(k=h;k<=no;k++){
			pi=pi+g_dev[k]*p_dev[ind_p2 + ((Nzz-idx)-k)*Nxx];
		}
		for(h=1;h<=ma_dev[idx];h++){
			cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		}
		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
		uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

		p_dev[ind_p1 + (Nzz-idx)*Nxx]=p_dev[ind_p1 + (Nzz-idx)*Nxx]+vel_dev[(Nzz-idx)*Nxx + idy]*aa;	
	}
	/*if (idx>0 && idx<Na && idy>=0 && idy<lim2){//lim2=(Nxx-1)-2*(Na-1)
		idy=idy+Na;

		int ind_p1=k1*Nzz*Nxx + idy;
		int ind_p2=k2*Nzz*Nxx + idy;
		int ind_p3=k2*Nzz*Nxx;
		int ind_Oz=idy;

		pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
		dl=pi;
		cf=0.0;
		for(h=1;h<=mn_dev[idx];h++){
			aa=p_dev[ind_p2 + ((Nzz-idx)-h)*Nxx] + p_dev[ind_p2 + ((Nzz-idx)+h)*Nxx];
			ak=p_dev[ind_p3 + (idy+h) + (Nzz-idx)*Nxx] + p_dev[ind_p3 + (idy-h) + (Nzz-idx)*Nxx];
			pi=pi+g_dev[h]*aa;
			dl=dl+g_dev[h]*ak;
			cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];	
		}
		for(k=h;k<=no;k++){
			pi=pi+g_dev[k]*(p_dev[ind_p2 + ((Nzz-idx)-k)*Nxx]);
			dl=dl+g_dev[k]*(p_dev[ind_p3 + (idy+k) + (Nzz-idx)*Nxx] + p_dev[ind_p3 + (idy-k) + (Nzz-idx)*Nxx]);
		}
		for(h=1;h<=ma_dev[idx];h++){
			cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		}
		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
		ak=aa+dl;
		uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

		p_dev[ind_p1 + (Nzz-idx)*Nxx]=vel_dev[(Nzz-idx)*Nxx + idy]*ak + 2.0f*p_dev[ind_p2 + (Nzz-idx)*Nxx] - 
			                  p_dev[ind_p1 + (Nzz-idx)*Nxx];
		
	}*/
	/*if (idx>0 && idx<Na && idy>=0 && idy<Na){
		idy=idy+(Nxx-Na+1);

		int ind_p1=k1*Nzz*Nxx + idy;
		int ind_p2=k2*Nzz*Nxx + idy;
		int ind_Oz=idy;

		pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
		cf=0.0;
		for(h=1;h<=mn_dev[idx];h++){
			pi=pi+g_dev[h]*(p_dev[ind_p2 + ((Nzz-idx)-h)*Nxx]+p_dev[ind_p2 + ((Nzz-idx)+h)*Nxx]);
			cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];
		}
		for(k=h;k<=no;k++){
			pi=pi+g_dev[k]*p_dev[ind_p2 + ((Nzz-idx)-k)*Nxx];
		}
		for(h=1;h<=ma_dev[idx];h++){
			cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		}	
		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
		uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];
		p_dev[ind_p1 + (Nzz-idx)*Nxx]=p_dev[ind_p1 + (Nzz-idx)*Nxx]+vel_dev[(Nzz-idx)*Nxx + idy]*aa;	
		//i*Nxx+j

	}*/
}

