#include "hip/hip_runtime.h"


__global__ void ParaOxOz(float *p_dev,float *Ox_dev,float *Oz_dev,float *Tt_dev,
											   //int *mn_dev,int *ma_dev,
											   float tx,float dh,
											   int N,int M,int no,int Na,int Nzz,int Nxx,int ms,int k1){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;
	float da,cf,ak,aa;
	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
		da=0.0;     cf=0.0;
		int ind_p=k1*Nzz*Nxx + idy*Nxx;
		int ind_Ox1=idx + idy*Na;
		int ind_Ox2=Nzz*Na + idx + idy*Na;

		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			
			da=da+d_dev[h]*(p_dev[ind_p + (idx+h)]-p_dev[ind_p + (idx-h)]);
			cf=cf+d_dev[h]*(p_dev[ind_p + ((Nxx-idx)+h)]-p_dev[ind_p + ((Nxx-idx)-h)]);
		}
		for(k=h;k<=no;k++){
			da=da+d_dev[k]*p_dev[ind_p + (idx+k)];
			cf=cf-d_dev[k]*p_dev[ind_p + ((Nxx-idx)-k)];
		}
		aa=tx*da + Tt_dev[idx]*Ox_dev[ind_Ox1];
		Ox_dev[ind_Ox1]=aa;
        ak=tx*cf + Tt_dev[idx]*Ox_dev[ind_Ox2];
		Ox_dev[ind_Ox2]=ak;

	}
	if (idx>0 && idx<Na && idy>0 && idy<Nxx){
		int ind_p=k1*Nzz*Nxx + idy;
		int ind_Oz=idy + idx*Nxx;
		cf=0.0;
		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			cf=cf+d_dev[h]*(p_dev[ind_p + (Nzz-idx+h)*Nxx] - p_dev[ind_p + (Nzz-idx-h)*Nxx]);
		}
		for(k=h;k<=no;k++){
			cf=cf-d_dev[k]*p_dev[ind_p + (Nzz-idx-k)*Nxx];
		}
		aa=tx*cf + Tt_dev[idx]*Oz_dev[ind_Oz];
		Oz_dev[ind_Oz]=aa;
	}
}
__global__ void ParaOxOz2(float *Ox_dev,float *Oz_dev,float *Tt_dev,
											   //int *mn_dev,int *ma_dev,
											   float tx,float dh,
											   int N,int M,int no,int Na,int Nzz,int Nxx,int ms,int k1){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;
	float da,cf,ak,aa;
	float val_p2_1;	float val_p2_2;	float val_p2_3;	float val_p2_4;	
	float val_p1_1;	float val_p1_2;	float val_p1_3;	float val_p1_4;

	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
		da=0.0;     cf=0.0;
		//int ind_p=k1*Nzz*Nxx + idy*Nxx;
		int ind_Ox1=idx + idy*Na;
		int ind_Ox2=Nzz*Na + idx + idy*Na;

		if(k1==0){
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p1_1=tex2D(tex_p1,(idx+h),idy);
				val_p1_2=tex2D(tex_p1,(idx-h),idy);
				val_p1_3=tex2D(tex_p1,((Nxx-idx)+h),idy);
				val_p1_4=tex2D(tex_p1,((Nxx-idx)-h),idy);	
				
				/*da=da+d_dev[h]*(p_dev[ind_p + (idx+h)]-p_dev[ind_p + (idx-h)]);
				cf=cf+d_dev[h]*(p_dev[ind_p + ((Nxx-idx)+h)]-p_dev[ind_p + ((Nxx-idx)-h)]);*/

				da=da+d_dev[h]*(val_p1_1 - val_p1_2);
				cf=cf+d_dev[h]*(val_p1_3 - val_p1_4);
			}
			for(k=h;k<=no;k++){
				val_p1_1=tex2D(tex_p1,(idx+k),idy);
				val_p1_2=tex2D(tex_p1,((Nxx-idx)-k),idy);

				/*da=da+d_dev[k]*p_dev[ind_p + (idx+k)];
			cf=cf-d_dev[k]*p_dev[ind_p + ((Nxx-idx)-k)];*/
				
				da=da+d_dev[k]*val_p1_1;
				cf=cf-d_dev[k]*val_p1_2;
			}	
			aa=tx*da + Tt_dev[idx]*Ox_dev[ind_Ox1];
			Ox_dev[ind_Ox1]=aa;
			ak=tx*cf + Tt_dev[idx]*Ox_dev[ind_Ox2];
			Ox_dev[ind_Ox2]=ak;	
		}
		else{
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p2_1=tex2D(tex_p2,(idx+h),idy);
				val_p2_2=tex2D(tex_p2,(idx-h),idy);
				val_p2_3=tex2D(tex_p2,((Nxx-idx)+h),idy);
				val_p2_4=tex2D(tex_p2,((Nxx-idx)-h),idy);	

				da=da+d_dev[h]*(val_p2_1 - val_p2_2);
				cf=cf+d_dev[h]*(val_p2_3 - val_p2_4);
			}
			for(k=h;k<=no;k++){
				val_p2_1=tex2D(tex_p2,(idx+k),idy);
				val_p2_2=tex2D(tex_p2,((Nxx-idx)-k),idy);
				
				da=da+d_dev[k]*val_p2_1;
				cf=cf-d_dev[k]*val_p2_2;
			}	
			aa=tx*da + Tt_dev[idx]*Ox_dev[ind_Ox1];
			Ox_dev[ind_Ox1]=aa;
			ak=tx*cf + Tt_dev[idx]*Ox_dev[ind_Ox2];
			Ox_dev[ind_Ox2]=ak;	
		}
	}
	if (idx>0 && idx<Na && idy>0 && idy<Nxx){
		//int ind_p=k1*Nzz*Nxx + idy;
		int ind_Oz=idy + idx*Nxx;
		cf=0.0;

		if(k1==0){
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p1_1=tex2D(tex_p1,idy,(Nzz-idx+h));
				val_p1_2=tex2D(tex_p1,idy,(Nzz-idx-h));

				cf=cf+d_dev[h]*(val_p1_1 - val_p1_2);
			}
			for(k=h;k<=no;k++){
				val_p1_1=tex2D(tex_p1,idy,(Nzz-idx-k));
				cf=cf-d_dev[k]*val_p1_1;
			}
			aa=tx*cf + Tt_dev[idx]*Oz_dev[ind_Oz];
			Oz_dev[ind_Oz]=aa;		
		}
		else{
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p2_1=tex2D(tex_p2,idy,(Nzz-idx+h));
				val_p2_2=tex2D(tex_p2,idy,(Nzz-idx-h));

				cf=cf+d_dev[h]*(val_p2_1 - val_p2_2);
			}
			for(k=h;k<=no;k++){
				val_p2_1=tex2D(tex_p2,idy,(Nzz-idx-k));
				cf=cf-d_dev[k]*val_p2_1;
			}
			aa=tx*cf + Tt_dev[idx]*Oz_dev[ind_Oz];
			Oz_dev[ind_Oz]=aa;		
		}
	}
}
