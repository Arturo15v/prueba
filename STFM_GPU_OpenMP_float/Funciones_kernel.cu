#include "hip/hip_runtime.h"
__constant__ float g_dev[6]; //ms=no+1
__constant__ float d_dev[6]; //no=4
texture<float, 2> tex_vel;
texture<int, 1> tex_mn;
texture<int, 1> tex_ma;

texture<float, 2> tex_p1;
texture<float, 2> tex_p2;


__global__ void Calcular_Tn(float *Tn_dev,float pi,float dl,float da,float cf,int N){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>0 && idx<N){
		Tn_dev[idx]=cf*pow((pi/dl),da);
	}
}

__global__ void Calcular_Tk_Tt(float *Tk_dev,float *Tt_dev,float *Tn_dev,float aa,float dl,float dt,int N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float cf;
	if (idx>0 && idx<N){
		cf=Tn_dev[idx] + aa;//No acum...
        Tk_dev[idx]=cf*dl;//P
        Tt_dev[idx]=(1.0f - dt*cf);//P
	}

}

__global__ void Inicializar_p(float *p_dev,int Nzz,int Nxx,int Dep){
	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;
	int idz=blockIdx.z * blockDim.z + threadIdx.z;
	
	if (idx>0 && idx<Nzz && idy>0 && idy<Nxx && idz<Dep){
		p_dev[idz*Nzz*Nxx + idy + idx*Nxx]=0.0;//Error idy*Nzz+idx
	}

}
__global__ void Inicializar_Ox_ux(float *Ox_dev,float *ux_dev,int Nzz,int Na,int Dep){
	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;
	int idz=blockIdx.z * blockDim.z + threadIdx.z;
	
	if (idx>0 && idx<Nzz && idy>0 && idy<Na && idz<Dep){
		Ox_dev[idz*Nzz*Na + idy + idx*Na]=0.0;
		ux_dev[idz*Nzz*Na + idy + idx*Na]=0.0;
	}

}
__global__ void Inicializar_uz_Oz(float *uz_dev,float *Oz_dev,int Na,int Nxx){
	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;
	if (idx>0 && idx<Na && idy>0 && idy<Nxx){
		Oz_dev[idy + idx*Nxx]=0.0;
		uz_dev[idy + idx*Nxx]=0.0;
	}


}


/*__global__ void Fuentes_multiples(float *p_dev,float *ricker_dev,int izs,int i_time,int ixs,int ibs,int N,int Nzz,int Nxx){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float val_ricker[2];
	val_ricker[0]=ricker_dev[i_time];
	val_ricker[1]=ricker_dev[i_time+1];
	__syncthreads();

	if(idx>=0 && idx<N-1){
		p_dev[izs*Nxx+(idx+ixs)]=val_ricker[0];
		p_dev[Nzz*Nxx+ izs*Nxx + (idx+ixs)]=val_ricker[1];
	}
}*/
__global__ void Fuentes_multiples(float *p_dev,float ricker_itime1,float ricker_itime2,int izs,int ixs,int ibs,int N,int Nzz,int Nxx){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;

	if(idx>=0 && idx<N-1){
		p_dev[izs*Nxx+(idx+ixs)]=ricker_itime1;
		p_dev[Nzz*Nxx+ izs*Nxx + (idx+ixs)]=ricker_itime2;
	}
}




__global__ void Fuentes_multiples2(float *p_dev,float *ricker_dev,int izs,int i_time,int ixs,int ibs,int N,int Nzz,int Nxx,int k1){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float val_ricker[1];
	val_ricker[0]=ricker_dev[i_time];
	__syncthreads();

	if(idx>=0 && idx<N-1){
		p_dev[k1*Nzz*Nxx + izs*Nxx + (idx+ixs)]=val_ricker[0];
	}
}

__global__ void Fuentes_multiples3(float *p_dev,float ricker_itime,int izs,int ixs,int ibs,int N,int Nzz,int Nxx,int k1){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;

	if(idx>=0 && idx<N-1){
		p_dev[k1*Nzz*Nxx + izs*Nxx + (idx+ixs)]=ricker_itime;
	}
}






__global__ void Calcular_vecinos(int *mr_dev,int *ml_dev,int *nr_dev,int *nl_dev,int *nb_dev,
								 int *mn_dev,int *ma_dev,int no,int Na,int N){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int h,k;
	if(idx>0 && idx<Na){
		//para ux en n+1
		for(h=1;h<=mn_dev[idx];h++){
			nl_dev[(idx-1)*no+h]=idx-h;  nr_dev[(idx-1)*no+h]=idx+h;
			ml_dev[(idx-1)*no+h]=N-h;  mr_dev[(idx-1)*no+h]=N+h;	
		}
		for(k=h;k<no;k++){
			nr_dev[(idx-1)*no+k]=idx+k;  ml_dev[(idx-1)*no+k]=N-k;
		}
		for(h=1;h<=ma_dev[idx];h++){
			nb_dev[(idx-1)*no+h]=idx+h;
		}
	}
}
__global__ void Calcular_funciones_absorbentes1(float *p_dev,float *Ox_dev,float *ux_dev,float *Tn_dev,float *Tk_dev,
											   //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
											   float tx2,float dh,
											   int N,int no,int Na,int Nzz,int Nxx,int ms,int k1){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	float dl,da,pi,cf,ak,aa;
	int h,k;

	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
			int ind_p=k1*Nzz*Nxx+idy*Nxx;
			int ind_Ox=Nzz*Na + idy*Na;
			int ind_ux=idx + idy*Na;
			int ind_ux2=Nzz*Na + idx + idy*Na;
			dl=g_dev[ms]*p_dev[ind_p + idx]; 
			pi=g_dev[ms]*p_dev[ind_p + (Nxx-idx)];
			da=0.0;                  cf=0.0;
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				dl=dl+g_dev[h]*(p_dev[ind_p + (idx-h)]+p_dev[ind_p + (idx+h)]);
				da=da-d_dev[h]*Ox_dev[(idx-h) + idy*Na]*Tn_dev[(idx-h)];
				pi=pi+g_dev[h]*(p_dev[ind_p + ((Nxx-idx)-h)]+p_dev[ind_p + ((Nxx-idx)+h)]);
				cf=cf+d_dev[h]*Ox_dev[ind_Ox + (idx-h)]*Tn_dev[(idx-h)];	
			}
			for(k=h;k<=no;k++){
				dl=dl+g_dev[k]*p_dev[ind_p + (idx+k)];
				pi=pi+g_dev[k]*p_dev[ind_p + ((Nxx-idx)-k)];
			}
			int val_ma=tex1D(tex_ma,idx);
			for(h=1;h<=val_ma;h++){
				//da=da+d_dev[h]*Ox_dev[(idx+h) + idy*Nxx]*Tn_dev[(idx+h)]; //Error2 es Na en vez de Nxx
				da=da+d_dev[h]*Ox_dev[(idx+h) + idy*Na]*Tn_dev[(idx+h)];
				cf=cf-d_dev[h]*Ox_dev[ind_Ox + (idx+h)]*Tn_dev[(idx+h)];
			}
			aa=dl - da*dh - Tk_dev[idx]*ux_dev[ind_ux];
			ux_dev[ind_ux]=tx2*aa+ux_dev[ind_ux];
			ak=pi - cf*dh - Tk_dev[idx]*ux_dev[ind_ux2];
			ux_dev[ind_ux2]=tx2*ak+ux_dev[ind_ux2];		
	}
}
__global__ void Calcular_funciones_absorbentes2(float *p_dev,float *Oz_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,
											   //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
											   float tx2,float dh,
											   int M,int no,int Na,int Nzz,int Nxx,int ms,int k1){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float pi,cf,aa;

	if (idx>0 && idx<Na && idy>0 && idy<Nxx){
	
		int ind_p=k1*Nzz*Nxx + idy;
		//int ind_Oz=idy;
		pi=g_dev[ms]*p_dev[ind_p + (Nzz-idx)*Nxx];
		cf=0.0;
		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			pi=pi+g_dev[h]*(p_dev[ind_p + ((Nzz-idx)-h)*Nxx]+p_dev[ind_p + ((Nzz-idx)+h)*Nxx]);
			cf=cf+d_dev[h]*Oz_dev[idy + (idx-h)*Nxx]*Tn_dev[(idx-h)];	
			
		}
		for(k=h;k<=no;k++)
			pi=pi+g_dev[k]*p_dev[ind_p + ((Nzz-idx)-k)*Nxx];
		int val_ma=tex1D(tex_ma,idx);
		for(h=1;h<=val_ma;h++)
			cf=cf-d_dev[h]*Oz_dev[idy + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		

		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[idy + idx*Nxx];
		uz_dev[idy + idx*Nxx]=tx2*aa+uz_dev[idy + idx*Nxx];

	}
}

__global__ void ParaOx(float *p_dev,float *Ox_dev,float *Tt_dev,
											   //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
											   float tx,float dh,
											   int N,int no,int Na,int Nzz,int Nxx,int ms,int k1){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float da,cf,ak,aa;
	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
		da=0.0;     cf=0.0;
		int ind_p=k1*Nzz*Nxx + idy*Nxx;
		int ind_Ox1=idx + idy*Na;
		int ind_Ox2=Nzz*Na + idx + idy*Na;

		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			
			da=da+d_dev[h]*(p_dev[ind_p + (idx+h)]-p_dev[ind_p + (idx-h)]);
			cf=cf+d_dev[h]*(p_dev[ind_p + ((Nxx-idx)+h)]-p_dev[ind_p + ((Nxx-idx)-h)]);
		}
		for(k=h;k<=no;k++){
			da=da+d_dev[k]*p_dev[ind_p + (idx+k)];
			cf=cf-d_dev[k]*p_dev[ind_p + ((Nxx-idx)-k)];
		}
		aa=tx*da + Tt_dev[idx]*Ox_dev[ind_Ox1];
		Ox_dev[ind_Ox1]=aa;
        ak=tx*cf + Tt_dev[idx]*Ox_dev[ind_Ox2];
		Ox_dev[ind_Ox2]=ak;

	}
}

__global__ void ParaOz(float *p_dev,float *Oz_dev,float *Tt_dev,
											   //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
											   float tx,float dh,
											   int M,int no,int Na,int Nzz,int Nxx,int ms,int k1){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float cf,aa;

	if (idx>0 && idx<Na && idy>0 && idy<Nxx){
		int ind_p=k1*Nzz*Nxx + idy;
		int ind_Oz=idy + idx*Nxx;
		cf=0.0;
		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			cf=cf+d_dev[h]*(p_dev[ind_p + (Nzz-idx+h)*Nxx] - p_dev[ind_p + (Nzz-idx-h)*Nxx]);
		}
		for(k=h;k<=no;k++){
			cf=cf-d_dev[k]*p_dev[ind_p + (Nzz-idx-k)*Nxx];
		}
		aa=tx*cf + Tt_dev[idx]*Oz_dev[ind_Oz];
		Oz_dev[ind_Oz]=aa;
	}
}

__global__ void EcDominioReal(float *p_dev,float *Oz_dev,float *Tt_dev,//float *vel_dev,
											   //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
											   float tx,float dh,
											   int M,int N,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;
	float dl,da;

	if (idx>0 && idx<=Nzz-Na && idy>=0 && idy<=N-M){
		idy=idy+M;	
		int ind_p1=k2*Nzz*Nxx;
		int ind_p2=k1*Nzz*Nxx;
		int ind_idxNxx=idx*Nxx;
		int ind_p3=ind_p2 + idy + ind_idxNxx;
		int ind_p4=idy + ind_idxNxx;

		dl=2.0f*g_dev[ms]*p_dev[ind_p1 + idx*Nxx + (idy)];
		int val_mn=tex1D(tex_mn,idx);
		//for(k=1;k<=mn_dev[idx];k++){
		for(k=1;k<=val_mn;k++){
			da=p_dev[ind_p1 + (idy-k) + idx*Nxx]   + p_dev[ind_p1 + (idy+k) + idx*Nxx] +
			   p_dev[ind_p1 + (idy) + (idx-k)*Nxx] + p_dev[ind_p1 + idy + (idx+k)*Nxx];
			dl=dl+g_dev[k]*da;
		}
		for(h=k;h<=no;h++){
			da=p_dev[ind_p1 + (idy-h) + idx*Nxx] + p_dev[ind_p1 + (idy+h) + idx*Nxx] + 
			   p_dev[ind_p1 + idy + (idx+h)*Nxx];
			dl=dl+g_dev[h]*da;
		}
		/*
		p_dev[ind_p2 + ((idy)) + idx*Nxx] = 2.0f*p_dev[ind_p1 + ((idy)) + idx*Nxx] -
			                                p_dev[ind_p2 + ((idy)) + idx*Nxx] + vel_dev[idx*Nxx + (idy)]*dl;
		*/
		//p_dev[ind_p3] = 2.0f*p_dev[ind_p1 + ind_p4]-p_dev[ind_p3] + vel_dev[ind_p4]*dl;

		float vel=tex2D(tex_vel,(float)idy,(float)idx);
		p_dev[ind_p3] =2.0f*p_dev[ind_p1 + ind_p4]-p_dev[ind_p3] + vel*dl;

		/*
		float val1=2*p_dev[ind_p1 + ind_p4];
		float val2=val1-p_dev[ind_p3];
		//float val3=vel_dev[ind_p4]*dl;
		p_dev[ind_p3] = val2;*/
		
	}
}
__global__ void EcDominioReal2(float *p_dev,float *Oz_dev,float *Tt_dev,//float *vel_dev,
											   //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
											   float tx,float dh,
											   int M,int N,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;
	float dl,da;
	float val_p1,val_p2;
	float val_p2_1;	float val_p2_2;	float val_p2_3;	float val_p2_4;	
	float val_p1_1;	float val_p1_2;	float val_p1_3;	float val_p1_4;

	if (idx>0 && idx<=Nzz-Na && idy>=0 && idy<=N-M){
		idy=idy+M;	
		//int ind_p1=k2*Nzz*Nxx;
		int ind_p2=k1*Nzz*Nxx;
		int ind_idxNxx=idx*Nxx;
		int ind_p3=ind_p2 + idy + ind_idxNxx;
		//int ind_p4=idy + ind_idxNxx;

		float vel=tex2D(tex_vel,idy,idx);
		if(k1==0){
			val_p2=tex2D(tex_p2,idy,idx);
			//dl=2.0f*g_dev[ms]*p_dev[ind_p1 + idx*Nxx + (idy)];
			dl=2.0f*g_dev[ms]*val_p2;
			int val_mn=tex1D(tex_mn,idx);
			//for(k=1;k<=mn_dev[idx];k++){
			for(k=1;k<=val_mn;k++){

				val_p2_1=tex2D(tex_p2,(idy-k),idx);
				val_p2_2=tex2D(tex_p2,(idy+k),idx);
				val_p2_3=tex2D(tex_p2,idy,(idx-k));
				val_p2_4=tex2D(tex_p2,idy,(idx+k));

				da=val_p2_1 + val_p2_2 + val_p2_3 + val_p2_4;
				
				//da=p_dev[ind_p1 + (idy-k) + idx*Nxx]   + p_dev[ind_p1 + (idy+k) + idx*Nxx] +
				  // p_dev[ind_p1 + (idy) + (idx-k)*Nxx] + p_dev[ind_p1 + idy + (idx+k)*Nxx];
				dl=dl+g_dev[k]*da;
			}
			for(h=k;h<=no;h++){
				val_p2_1=tex2D(tex_p2,(idy-h),idx);
				val_p2_2=tex2D(tex_p2,(idy+h),idx);
				val_p2_3=tex2D(tex_p2,idy,(idx+h));
				
				da=val_p2_1 + val_p2_2 + val_p2_3;
				//da=p_dev[ind_p1 + (idy-h) + idx*Nxx] + p_dev[ind_p1 + (idy+h) + idx*Nxx] + 
				  // p_dev[ind_p1 + idy + (idx+h)*Nxx];
				dl=dl+g_dev[h]*da;
			}
			
			val_p1=tex2D(tex_p1,idy,idx);
			//p_dev[ind_p3] =2.0f*p_dev[ind_p1 + ind_p4]-p_dev[ind_p3] + vel*dl;
			p_dev[ind_p3] =2.0f*val_p2 - val_p1 + vel*dl;
		}
		else{
			val_p1=tex2D(tex_p1,idy,idx);
			//dl=2.0f*g_dev[ms]*p_dev[ind_p1 + idx*Nxx + (idy)];
			dl=2.0f*g_dev[ms]*val_p1;
			int val_mn=tex1D(tex_mn,idx);
			//for(k=1;k<=mn_dev[idx];k++){
			for(k=1;k<=val_mn;k++){

				val_p1_1=tex2D(tex_p1,(idy-k),idx);
				val_p1_2=tex2D(tex_p1,(idy+k),idx);
				val_p1_3=tex2D(tex_p1,idy,(idx-k));
				val_p1_4=tex2D(tex_p1,idy,(idx+k));

				da=val_p1_1 + val_p1_2 + val_p1_3 + val_p1_4;
				
				//da=p_dev[ind_p1 + (idy-k) + idx*Nxx]   + p_dev[ind_p1 + (idy+k) + idx*Nxx] +
				  // p_dev[ind_p1 + (idy) + (idx-k)*Nxx] + p_dev[ind_p1 + idy + (idx+k)*Nxx];
				dl=dl+g_dev[k]*da;
			}
			for(h=k;h<=no;h++){
				val_p1_1=tex2D(tex_p1,(idy-h),idx);
				val_p1_2=tex2D(tex_p1,(idy+h),idx);
				val_p1_3=tex2D(tex_p1,idy,(idx+h));
				
				da=val_p1_1 + val_p1_2 + val_p1_3;
				//da=p_dev[ind_p1 + (idy-h) + idx*Nxx] + p_dev[ind_p1 + (idy+h) + idx*Nxx] + 
				  // p_dev[ind_p1 + idy + (idx+h)*Nxx];
				dl=dl+g_dev[h]*da;
			}
			
			val_p2=tex2D(tex_p2,idy,idx);
			//p_dev[ind_p3] =2.0f*p_dev[ind_p1 + ind_p4]-p_dev[ind_p3] + vel*dl;
			p_dev[ind_p3] =2.0f*val_p1 - val_p2 + vel*dl;
		}
	}
}

__global__ void Para_uxp(float *p_dev,float *Ox_dev,float *ux_dev,float *Tn_dev,float *Tk_dev,float *vel_dev,
						  //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						  float tx2,float dh,
						  int N,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	float dl,da,pi,cf,aa;
	int h,k;

	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
			int ind_p=k2*Nzz*Nxx+idy*Nxx;
			int ind_Ox=Nzz*Na+ idy*Na;
			int ind_ux=idx + idy*Na;
			int ind_ux2=Nzz*Na + idx + idy*Na;
			int ind_p2=k1*Nzz*Nxx + idx + idy*Nxx;
			int ind_p3=k1*Nzz*Nxx+idy*Nxx;

			dl=g_dev[ms]*p_dev[ind_p + idx];       
			pi=g_dev[ms]*p_dev[ind_p + (Nxx-idx)];
			da=0.0;                   cf=0.0;
			
			int val_mn=tex1D(tex_mn,idx);
			//for(h=1;h<=mn_dev[idx];h++){
			for(h=1;h<=val_mn;h++){
				dl=dl+g_dev[h]*(p_dev[ind_p + (idx-h)]+p_dev[ind_p + (idx+h)]);
				da=da-d_dev[h]*Ox_dev[(idx-h) + idy*Na]*Tn_dev[(idx-h)];
				pi=pi+g_dev[h]*(p_dev[ind_p + ((Nxx-idx)-h)]+p_dev[ind_p + ((Nxx-idx)+h)]);
				cf=cf+d_dev[h]*Ox_dev[ind_Ox + (idx-h)]*Tn_dev[(idx-h)];	
			}
			for(k=h;k<=no;k++){
				dl=dl+g_dev[k]*p_dev[ind_p + (idx+k)];
				pi=pi+g_dev[k]*p_dev[ind_p + ((Nxx-idx)-k)];
			}
			int val_ma=tex1D(tex_ma,idx);
			for(h=1;h<=val_ma;h++){
				da=da+d_dev[h]*Ox_dev[(idx+h) + idy*Na]*Tn_dev[(idx+h)];
				cf=cf-d_dev[h]*Ox_dev[ind_Ox + (idx+h)]*Tn_dev[(idx+h)];
			}

			/*float vel=tex2D(tex_vel,(float)idx+0.5,(float)idy+0.5);
			float vel2=tex2D(tex_vel,(float)(Nxx-idx)+0.5,(float)idy+0.5);
*/
			aa=dl - da*dh - Tk_dev[idx]*ux_dev[ind_ux];
			ux_dev[ind_ux]=tx2*aa+ux_dev[ind_ux];
			p_dev[ind_p2]=vel_dev[idy*Nxx + idx]*aa+ 2.0f*p_dev[ind_p + idx] - p_dev[ind_p2];
			//p_dev[ind_p2]=vel*aa+ 2.0f*p_dev[ind_p + idx] - p_dev[ind_p2];
			aa=pi - cf*dh - Tk_dev[idx]*ux_dev[ind_ux2];
			ux_dev[ind_ux2]=tx2*aa+ux_dev[ind_ux2];
			p_dev[ind_p3 + (Nxx-idx)]=vel_dev[idy*Nxx + (Nxx-idx)]*aa + 
			//p_dev[ind_p3 + (Nxx-idx)]=vel2*aa + 
				                          2.0f*p_dev[ind_p + (Nxx-idx)] - p_dev[ind_p3 + (Nxx-idx)];

	}
}
__global__ void Para_uxp2(float *p_dev,float *Ox_dev,float *ux_dev,float *Tn_dev,float *Tk_dev,//float *vel_dev,
						  //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						  float tx2,float dh,
						  int N,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	float dl,da,pi,cf,aa;
	int h,k;

	float val_p1,val_p2;
	float val_p2_1;	float val_p2_2;	float val_p2_3;	float val_p2_4;	
	float val_p1_1;	float val_p1_2;	float val_p1_3;	float val_p1_4;

	if (idx>0 && idx<Na && idy>0 && idy<Nzz){
			//int ind_p=k2*Nzz*Nxx+idy*Nxx;
			int ind_Ox=Nzz*Na+ idy*Na;
			int ind_ux=idx + idy*Na;
			int ind_ux2=Nzz*Na + idx + idy*Na;
			int ind_p2=k1*Nzz*Nxx + idx + idy*Nxx;
			int ind_p3=k1*Nzz*Nxx+idy*Nxx;
			
			
			float vel=tex2D(tex_vel,idx,idy);
			float vel2=tex2D(tex_vel,(Nxx-idx),idy);

			if(k1==0){

				val_p2=tex2D(tex_p2,idx,idy);
				val_p2_1=tex2D(tex_p2,(Nxx-idx),idy);

				//dl=g_dev[ms]*p_dev[ind_p + idx];       
				dl=g_dev[ms]*val_p2;       
				//pi=g_dev[ms]*p_dev[ind_p + (Nxx-idx)];
				pi=g_dev[ms]*val_p2_1;
				da=0.0;                   cf=0.0;

				int val_mn=tex1D(tex_mn,idx);
				//for(h=1;h<=mn_dev[idx];h++){
				for(h=1;h<=val_mn;h++){
					val_p2_1=tex2D(tex_p2,(idx-h),idy);	
					val_p2_2=tex2D(tex_p2,(idx+h),idy);	
					val_p2_3=tex2D(tex_p2,((Nxx-idx)-h),idy);	
					val_p2_4=tex2D(tex_p2,((Nxx-idx)+h),idy);	

					dl=dl+g_dev[h]*(val_p2_1+val_p2_2);
					da=da-d_dev[h]*Ox_dev[(idx-h) + idy*Na]*Tn_dev[(idx-h)];
					pi=pi+g_dev[h]*(val_p2_3+val_p2_4);
					cf=cf+d_dev[h]*Ox_dev[ind_Ox + (idx-h)]*Tn_dev[(idx-h)];	
				}
				for(k=h;k<=no;k++){
					val_p2_1=tex2D(tex_p2,(idx+k),idy);	
					val_p2_2=tex2D(tex_p2,((Nxx-idx)-k),idy);	
					dl=dl+g_dev[k]*val_p2_1;
					pi=pi+g_dev[k]*val_p2_2;
				}
				int val_ma=tex1D(tex_ma,idx);
				for(h=1;h<=val_ma;h++){
					da=da+d_dev[h]*Ox_dev[(idx+h) + idy*Na]*Tn_dev[(idx+h)];
					cf=cf-d_dev[h]*Ox_dev[ind_Ox + (idx+h)]*Tn_dev[(idx+h)];
				}
				aa=dl - da*dh - Tk_dev[idx]*ux_dev[ind_ux];
				ux_dev[ind_ux]=tx2*aa+ux_dev[ind_ux];
				
				val_p1=tex2D(tex_p1,idx,idy);
				p_dev[ind_p2]=vel*aa+ 2.0f*val_p2 - val_p1;

				aa=pi - cf*dh - Tk_dev[idx]*ux_dev[ind_ux2];
				ux_dev[ind_ux2]=tx2*aa+ux_dev[ind_ux2];
				
				val_p2_1=tex2D(tex_p2,(Nxx-idx),idy);
				val_p1_1=tex2D(tex_p1,(Nxx-idx),idy);

				p_dev[ind_p3 + (Nxx-idx)]=vel2*aa + 2.0f*val_p2_1 - val_p1_1;	
			}
			else{
				val_p1=tex2D(tex_p1,idx,idy);
				val_p1_1=tex2D(tex_p1,(Nxx-idx),idy);

				//dl=g_dev[ms]*p_dev[ind_p + idx];       
				dl=g_dev[ms]*val_p1;       
				//pi=g_dev[ms]*p_dev[ind_p + (Nxx-idx)];
				pi=g_dev[ms]*val_p1_1;
				da=0.0;                   cf=0.0;

				int val_mn=tex1D(tex_mn,idx);
				//for(h=1;h<=mn_dev[idx];h++){
				for(h=1;h<=val_mn;h++){
					val_p1_1=tex2D(tex_p1,(idx-h),idy);	
					val_p1_2=tex2D(tex_p1,(idx+h),idy);	
					val_p1_3=tex2D(tex_p1,((Nxx-idx)-h),idy);	
					val_p1_4=tex2D(tex_p1,((Nxx-idx)+h),idy);	

					dl=dl+g_dev[h]*(val_p1_1+val_p1_2);
					da=da-d_dev[h]*Ox_dev[(idx-h) + idy*Na]*Tn_dev[(idx-h)];
					pi=pi+g_dev[h]*(val_p1_3+val_p1_4);
					cf=cf+d_dev[h]*Ox_dev[ind_Ox + (idx-h)]*Tn_dev[(idx-h)];	
				}
				for(k=h;k<=no;k++){
					val_p1_1=tex2D(tex_p1,(idx+k),idy);	
					val_p1_2=tex2D(tex_p1,((Nxx-idx)-k),idy);	
					dl=dl+g_dev[k]*val_p1_1;
					pi=pi+g_dev[k]*val_p1_2;
				}
				int val_ma=tex1D(tex_ma,idx);
				for(h=1;h<=val_ma;h++){
					da=da+d_dev[h]*Ox_dev[(idx+h) + idy*Na]*Tn_dev[(idx+h)];
					cf=cf-d_dev[h]*Ox_dev[ind_Ox + (idx+h)]*Tn_dev[(idx+h)];
				}
				aa=dl - da*dh - Tk_dev[idx]*ux_dev[ind_ux];
				ux_dev[ind_ux]=tx2*aa+ux_dev[ind_ux];
				
				val_p2=tex2D(tex_p2,idx,idy);
				p_dev[ind_p2]=vel*aa+ 2.0f*val_p1 - val_p2;

				aa=pi - cf*dh - Tk_dev[idx]*ux_dev[ind_ux2];
				ux_dev[ind_ux2]=tx2*aa+ux_dev[ind_ux2];
				
				val_p1_1=tex2D(tex_p1,(Nxx-idx),idy);
				val_p2_1=tex2D(tex_p2,(Nxx-idx),idy);

				p_dev[ind_p3 + (Nxx-idx)]=vel2*aa + 2.0f*val_p1_1 - val_p2_1;	
			}
	}
}


__global__ void Para_uzp(float *p_dev,float *Oz_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,float *vel_dev,
						 //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						 float tx2,float dh,
						 int M,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float pi,cf,aa;

	if (idx>0 && idx<Na && idy>0 && idy<Na){

		int ind_p1=k1*Nzz*Nxx + idy;
		int ind_p2=k2*Nzz*Nxx + idy;
		int ind_Oz=idy;

		pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
		cf=0.0;
		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
		//for(h=1;h<=mn_dev[idx];h++){
			pi=pi+g_dev[h]*(p_dev[ind_p2 + ((Nzz-idx)-h)*Nxx]+p_dev[ind_p2 + ((Nzz-idx)+h)*Nxx]);
			cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];

		}
		for(k=h;k<=no;k++){
			pi=pi+g_dev[k]*p_dev[ind_p2 + ((Nzz-idx)-k)*Nxx];
		}
		int val_ma=tex1D(tex_ma,idx);
		for(h=1;h<=val_ma;h++){
			cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		}
		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
		uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

		//float vel=tex2D(tex_vel,(float)idy+0.5,(float)(Nzz-idx)+0.5);
		p_dev[ind_p1 + (Nzz-idx)*Nxx]=p_dev[ind_p1 + (Nzz-idx)*Nxx]+vel_dev[(Nzz-idx)*Nxx + idy]*aa;	
		//p_dev[ind_p1 + (Nzz-idx)*Nxx]=p_dev[ind_p1 + (Nzz-idx)*Nxx]+vel*aa;	
	}
}
__global__ void Para_uzp2(float *p_dev,float *Oz_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,//float *vel_dev,
						 //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						 float tx2,float dh,
						 int M,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float pi,cf,aa;

	float val_p1,val_p2;
	float val_p2_1;	float val_p2_2;	
	float val_p1_1;	float val_p1_2;	
	if (idx>0 && idx<Na && idy>0 && idy<Na){
		int ind_p1=k1*Nzz*Nxx + idy;
		//int ind_p2=k2*Nzz*Nxx + idy;
		int ind_Oz=idy;
		float vel=tex2D(tex_vel,idy,(Nzz-idx));

		if(k1==0){
			val_p2=tex2D(tex_p2,idy,(Nzz-idx));
			//pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
			pi=g_dev[ms]*val_p2;
			cf=0.0;
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p2_1=tex2D(tex_p2,idy,((Nzz-idx)-h));
				val_p2_2=tex2D(tex_p2,idy,((Nzz-idx)+h));

				pi=pi+g_dev[h]*(val_p2_1+val_p2_2);
				cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];

			}
			for(k=h;k<=no;k++){
				val_p2_1=tex2D(tex_p2,idy,((Nzz-idx)-k));
				pi=pi+g_dev[k]*val_p2_1;
			}
			int val_ma=tex1D(tex_ma,idx);
			for(h=1;h<=val_ma;h++){
				cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
			}
			aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
			uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

			val_p1=tex2D(tex_p1,idy,(Nzz-idx));
			p_dev[ind_p1 + (Nzz-idx)*Nxx]=val_p1+vel*aa;	
		
		}
		else{
			val_p1=tex2D(tex_p1,idy,(Nzz-idx));
			//pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
			pi=g_dev[ms]*val_p1;
			cf=0.0;
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p1_1=tex2D(tex_p1,idy,((Nzz-idx)-h));
				val_p1_2=tex2D(tex_p1,idy,((Nzz-idx)+h));

				pi=pi+g_dev[h]*(val_p1_1+val_p1_2);
				cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];

			}
			for(k=h;k<=no;k++){
				val_p1_1=tex2D(tex_p1,idy,((Nzz-idx)-k));
				pi=pi+g_dev[k]*val_p1_1;
			}
			int val_ma=tex1D(tex_ma,idx);
			for(h=1;h<=val_ma;h++){
				cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
			}
			aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
			uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

			val_p2=tex2D(tex_p2,idy,(Nzz-idx));
			p_dev[ind_p1 + (Nzz-idx)*Nxx]=val_p2+vel*aa;	
		
		}
	}
}
__global__ void Enmedio_uz(float *p_dev,float *Oz_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,float *vel_dev,
						// int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						 float tx2,float dh,
						 int M,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2, int lim2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float pi,cf,aa,dl,ak;

	if (idx>0 && idx<Na && idy>=0 && idy<lim2){//lim2=(Nxx-1)-2*(Na-1)
		idy=idy+Na;

		int ind_p1=k1*Nzz*Nxx + idy;
		int ind_p2=k2*Nzz*Nxx + idy;
		int ind_p3=k2*Nzz*Nxx;
		int ind_Oz=idy;

		pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
		dl=pi;
		cf=0.0;
		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			aa=p_dev[ind_p2 + ((Nzz-idx)-h)*Nxx] + p_dev[ind_p2 + ((Nzz-idx)+h)*Nxx];
			ak=p_dev[ind_p3 + (idy+h) + (Nzz-idx)*Nxx] + p_dev[ind_p3 + (idy-h) + (Nzz-idx)*Nxx];
			pi=pi+g_dev[h]*aa;
			dl=dl+g_dev[h]*ak;
			cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];	
		}
		for(k=h;k<=no;k++){
			pi=pi+g_dev[k]*(p_dev[ind_p2 + ((Nzz-idx)-k)*Nxx]);
			dl=dl+g_dev[k]*(p_dev[ind_p3 + (idy+k) + (Nzz-idx)*Nxx] + p_dev[ind_p3 + (idy-k) + (Nzz-idx)*Nxx]);
		}
		int val_ma=tex1D(tex_ma,idx);
		for(h=1;h<=val_ma;h++){
			cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		}
		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
		ak=aa+dl;
		uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

		//float vel=tex2D(tex_vel,(float)idy+0.5,(float)(Nzz-idx)+0.5);
		
		p_dev[ind_p1 + (Nzz-idx)*Nxx]=vel_dev[(Nzz-idx)*Nxx + idy]*ak + 2.0f*p_dev[ind_p2 + (Nzz-idx)*Nxx] - 
		//p_dev[ind_p1 + (Nzz-idx)*Nxx]=vel*ak + 2.0f*p_dev[ind_p2 + (Nzz-idx)*Nxx] - 
			                  p_dev[ind_p1 + (Nzz-idx)*Nxx];
		
	}
}

__global__ void Enmedio_uz2(float *p_dev,float *Oz_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,
						 //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						 float tx2,float dh,
						 int M,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2, int lim2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float pi,cf,aa,dl,ak;
	float val_p1,val_p2;
	float val_p2_1;	float val_p2_2;	float val_p2_3;	float val_p2_4;	
	float val_p1_1;	float val_p1_2;	float val_p1_3;	float val_p1_4;

	if (idx>0 && idx<Na && idy>=0 && idy<lim2){//lim2=(Nxx-1)-2*(Na-1)
		idy=idy+Na;

		int ind_p1=k1*Nzz*Nxx + idy;
		//int ind_p2=k2*Nzz*Nxx + idy;
		//int ind_p3=k2*Nzz*Nxx;
		int ind_Oz=idy;
		float vel=tex2D(tex_vel,idy,(Nzz-idx));

		if(k1==0){
			val_p2=tex2D(tex_p2,idy,(Nzz-idx));

			//pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
			pi=g_dev[ms]*val_p2;
			dl=pi;
			cf=0.0;
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p2_1=tex2D(tex_p2,idy,((Nzz-idx)-h));
				val_p2_2=tex2D(tex_p2,idy,((Nzz-idx)+h));
				val_p2_3=tex2D(tex_p2,(idy+h),(Nzz-idx));
				val_p2_4=tex2D(tex_p2,(idy-h),(Nzz-idx));
				aa=val_p2_1 + val_p2_2;
				ak=val_p2_3 + val_p2_4;
				pi=pi+g_dev[h]*aa;
				dl=dl+g_dev[h]*ak;
				cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];	
			}
			for(k=h;k<=no;k++){
				val_p2_1=tex2D(tex_p2,idy,((Nzz-idx)-k));
				val_p2_2=tex2D(tex_p2,(idy+k),(Nzz-idx));
				val_p2_3=tex2D(tex_p2,(idy-k),(Nzz-idx));
				
				pi=pi+g_dev[k]*(val_p2_1);
				dl=dl+g_dev[k]*(val_p2_2 + val_p2_3);
			}
			int val_ma=tex1D(tex_ma,idx);
			for(h=1;h<=val_ma;h++){
				cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
			}
			aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
			ak=aa+dl;
			uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

			val_p1=tex2D(tex_p1,idy,(Nzz-idx));
			p_dev[ind_p1 + (Nzz-idx)*Nxx]=vel*ak + 2.0f*val_p2 - val_p1;
		}
		else{
			val_p1=tex2D(tex_p1,idy,(Nzz-idx));

			//pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
			pi=g_dev[ms]*val_p1;
			dl=pi;
			cf=0.0;
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				val_p1_1=tex2D(tex_p1,idy,((Nzz-idx)-h));
				val_p1_2=tex2D(tex_p1,idy,((Nzz-idx)+h));
				val_p1_3=tex2D(tex_p1,(idy+h),(Nzz-idx));
				val_p1_4=tex2D(tex_p1,(idy-h),(Nzz-idx));
				aa=val_p1_1 + val_p1_2;
				ak=val_p1_3 + val_p1_4;
				pi=pi+g_dev[h]*aa;
				dl=dl+g_dev[h]*ak;
				cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];	
			}
			for(k=h;k<=no;k++){
				val_p1_1=tex2D(tex_p1,idy,((Nzz-idx)-k));
				val_p1_2=tex2D(tex_p1,(idy+k),(Nzz-idx));
				val_p1_3=tex2D(tex_p1,(idy-k),(Nzz-idx));
				
				pi=pi+g_dev[k]*(val_p1_1);
				dl=dl+g_dev[k]*(val_p1_2 + val_p1_3);
			}
			int val_ma=tex1D(tex_ma,idx);
			for(h=1;h<=val_ma;h++){
				cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
			}
			aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
			ak=aa+dl;
			uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];

			val_p2=tex2D(tex_p2,idy,(Nzz-idx));
			p_dev[ind_p1 + (Nzz-idx)*Nxx]=vel*ak + 2.0f*val_p1 - val_p2;
		
		}
	}
}

__global__ void Final_uzp(float *p_dev,float *Oz_dev,float *uz_dev,float *Tn_dev,float *Tk_dev,float *vel_dev,
						 //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						 float tx2,float dh,
						 int M,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){


	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	int h,k;

	float pi,cf,aa;

	if (idx>0 && idx<Na && idy>=0 && idy<Na){
		idy=idy+(Nxx-Na+1);

		int ind_p1=k1*Nzz*Nxx + idy;
		int ind_p2=k2*Nzz*Nxx + idy;
		int ind_Oz=idy;

		pi=g_dev[ms]*p_dev[ind_p2 + (Nzz-idx)*Nxx];
		cf=0.0;
		int val_mn=tex1D(tex_mn,idx);
		for(h=1;h<=val_mn;h++){
			pi=pi+g_dev[h]*(p_dev[ind_p2 + ((Nzz-idx)-h)*Nxx]+p_dev[ind_p2 + ((Nzz-idx)+h)*Nxx]);
			cf=cf+d_dev[h]*Oz_dev[ind_Oz + (idx-h)*Nxx]*Tn_dev[(idx-h)];
		}
		for(k=h;k<=no;k++){
			pi=pi+g_dev[k]*p_dev[ind_p2 + ((Nzz-idx)-k)*Nxx];
		}
		int val_ma=tex1D(tex_ma,idx);
		for(h=1;h<=val_ma;h++){
			cf=cf-d_dev[h]*Oz_dev[ind_Oz + (idx+h)*Nxx]*Tn_dev[(idx+h)];
		}	
		aa=pi - cf*dh - Tk_dev[idx]*uz_dev[ind_Oz + idx*Nxx];
		uz_dev[ind_Oz + idx*Nxx]=tx2*aa+uz_dev[ind_Oz + idx*Nxx];
		
		//float vel=tex2D(tex_vel,(float)idy+0.5,(float)(Nzz-idx)+0.5);
		
		p_dev[ind_p1 + (Nzz-idx)*Nxx]=p_dev[ind_p1 + (Nzz-idx)*Nxx]+vel_dev[(Nzz-idx)*Nxx + idy]*aa;	
		//p_dev[ind_p1 + (Nzz-idx)*Nxx]=p_dev[ind_p1 + (Nzz-idx)*Nxx]+vel*aa;	
		//i*Nxx+j

	}
}

__global__ void ParteSuperior_ux(float *p_dev,float *Ox_dev,float *ux_dev,float *Tn_dev,float *Tk_dev,float *vel_dev,
						  //int *mn_dev,int *ma_dev,//float *g_dev,float *d_dev,
						  float tx2,float dh,
						  int N,int no,int Na,int Nzz,int Nxx,int ms,int k1,int k2){

	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;

	float dl,pi;
	int h,k;


	if (idx>0 && idx<=Nzz-Na && idy>0 && idy<Na){

			int ind_p1=k1*Nzz*Nxx + idy + idx*Nxx;
			int ind_p2=k2*Nzz*Nxx + idx*Nxx;
			int ind_p3=k2*Nzz*Nxx + idy;
			int ind_p4=k2*Nzz*Nxx + (Nxx-idy);
			int ind_p5=k1*Nzz*Nxx + (Nxx-idy) + idx*Nxx;

			dl=g_dev[ms]*p_dev[ind_p2 + idy];
			pi=g_dev[ms]*p_dev[ind_p2 + (Nxx-idy)];//Error1
			int val_mn=tex1D(tex_mn,idx);
			for(h=1;h<=val_mn;h++){
				dl=dl+g_dev[h]*(p_dev[ind_p3 + (idx-h)*Nxx]+p_dev[ind_p3 + (idx+h)*Nxx]);
				pi=pi+g_dev[h]*(p_dev[ind_p4 + (idx-h)*Nxx]+p_dev[ind_p4 + (idx+h)*Nxx]);
			}
			for(k=h;k<=no;k++){
				dl=dl+g_dev[k]*p_dev[ind_p3 + (idx+k)*Nxx];
				pi=pi+g_dev[k]*p_dev[ind_p4 + (idx+k)*Nxx];
			}

			/*float vel=tex2D(tex_vel,(float)idy+0.5,(float)idx+0.5);
			float vel2=tex2D(tex_vel,(float)(Nxx-idy)+0.5,(float)idx+0.5);*/
			p_dev[ind_p1]=p_dev[ind_p1]+vel_dev[idx*Nxx + idy]*dl;
			//p_dev[ind_p1]=p_dev[ind_p1]+vel*dl;
			p_dev[ind_p5]=p_dev[ind_p5]+vel_dev[idx*Nxx + (Nxx-idy)]*pi;
			//p_dev[ind_p5]=p_dev[ind_p5]+vel2*pi;
	}
}

__global__ void Pasar_Valores(float *p_dev2,float *p_dev,int N,int Nxx,int Na,int Nzz,int k1){
	
	int idx=blockIdx.x * blockDim.x + threadIdx.x;

	if(idx<N){
		p_dev2[idx]=p_dev[k1*Nzz*Nxx + Nxx + Na+idx];
		//p_host[(k1-1)*(Nzz+1)*(Nxx+1) + (Nxx+1) + j]
	}
}

__global__ void Pasar_Valores_vel(float *vel_dev,int Nzz,int Nxx){
	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	int idy=blockIdx.y * blockDim.y + threadIdx.y;
	if(idx>=0 && idx<Nzz && idy>=0 && idy<Nxx){
		vel_dev[idx*Nxx+idy]=tex2D(tex_vel,(float)idy,(float)idx);
	}
}

__global__ void Pasar_Valores_vel(int *mn_dev,int Nzz){
	int idx=blockIdx.x * blockDim.x + threadIdx.x;
	if(idx<Nzz){
		mn_dev[idx]=tex1D(tex_mn,idx);
	}
}