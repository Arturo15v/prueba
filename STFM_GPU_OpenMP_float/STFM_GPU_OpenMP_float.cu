#include "hip/hip_runtime.h"
// STFM_GPU_OpenMP_float.cpp : Defines the entry point for the console application.
//

//-------pasar a linux, correr unas cuantas iteraciones y realizar fwi sin kernells, comparar
/*
	Programa que resuelve la ecuaci�n de onda escalar en 2D (X-Z,t).
C   Se aplican fronteras absorbentes tipo PML.
C   La soluci�n de la ecuaci�n de onda se hace a trav�s del m�todo de
C   diferencias finitas de orden "no" en espacio y segundo orden en
C   tiempo con condiciones de Dirichlet P=0 en toda la frontera.
*/

// para ejecutar nvcc -o ejecutable.out programa.cu
// ejecutar con ./ejecutable.out
// para verificar tiempo nvprof ./ejecutable.out
//para revisar nucleoscuda es en usr/local/cuda-8.0/samples
//para reconocer device checar pagina 55
//

//-----Para compilar omp se usa nvcc -arch=sm_20 -Xcompiler -fopenmp -Igomp -o ejecutable.out programa.cu
//comentario al azat lol
#include <stdio.h>
#include <stdlib.h>
//#include <conio.h>
//---para conio.h se puede usar #include <curses.h> o <ncurses.h> pero se necesita descargar la libreria
#include <math.h>
#include "pidesist.h"

//#include <cutil.h>
//---para solucionar cutil se puede usar hip/hip_vector_types.h pero no sirve para operadores logicos
#include "Funciones_kernel.cu"
#include "FuncionesOxOz_kernel.cu"
//#include "Funciones_uxp_uzp_kernel.cu"

//#include <windows.h>
//---el windows.h hay que solucionarlo cambiando los comandos
#include <time.h>
//---para remplazar cuda safe call
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//OpenMP
#include <omp.h>

/*
implicit INTEGER (h-n)
implicit float PRECISION (a-g,o-z)
INTEGER,ALLOCATABLE,DIMENSION(:) :: nl,nr,nb,mn,ma,ml,mr

float PRECISION,ALLOCATABLE,DIMENSION(:) :: Tn,Tk,Tt,ricker,d,g
float PRECISION,ALLOCATABLE,DIMENSION(:,:) :: vel,uz,Oz
float PRECISION,ALLOCATABLE,DIMENSION(:,:,:) :: p,ux,Ox
*/



void Ajusta_profundidad(int Na,int no);
void Fuente_Ricker(float *ricker,int nrick,float fmid,float dt);
void Llenar_vect_enteros_pasos(int *mn,int *ma,int no,int Nzz,int Na);
void Estimando_vect_orden(float *g,float *d,int no);
void Modelo_velocidades(float **vel,int Nx,int Nz);
float Estabilidad_numerica(float **vel,int Nx,int Nz,float dt,float dh,float *vmax);
void Modif_vel_Expandir_malla(float **vel,int Nx,int Nz,int Na,int Nxx,int Nzz,float tx);
void Calculo_Ec_Onda(float ***p,float ***Ox,float ***ux,
					 float **uz,float **Oz,float *mvel,float **vel,
					 float *Tn,float *Tk,float *Tt,float *ricker,float *g,float *d,
					 int *mr,int *ml,int *nr,int *nl,int *nb,int *mn,int *ma,
					 float dt,float dh,float vmax,float da,float aa,float tx,float fmid,
					 int ixs,int nixs,int Na,int Nxx,int Nzz,int no,int izs,int ms,int nt,int ibs,int nrick);

void Guardar_Info_Var(float ***p,float ***Ox,float ***ux,float **Oz,float **uz,int Nzz,int Nxx,int Na);
void Pasar_Inf_Var(float ***p,float ***Ox,float ***ux,float **Oz,float **uz,
				   float *p_host,float *Ox_host,float *ux_host,float *Oz_host,float *uz_host,
				   int Nzz,int Nxx,int Na);

//Divisi�n techo.
int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

hipArray* cu_array_vel;
hipArray* cu_array_p1;
hipArray* cu_array_p2;
hipArray* cu_array_mn;
hipArray* cu_array_ma;

//int omp_get_num_procs(void);


int main (int argc, char **argv){

/*	----------------TEMPORALMENTE BLOQUEADO MIENTRAS SE ARREGLA EL SYSTEMTIME
	SYSTEMTIME st_ini,st_fin;
	
	GetSystemTime(&st_ini);
*/
	int num_proc=omp_get_num_procs();
	printf("\nNumero de procesadores: %d\n",num_proc);
	
	omp_set_num_threads(num_proc);//num_proc
    
/*
C----------------------------------------------------------------------
C     N�mero de orden de las diferencias en espacio
C             no=1  2do orden
C             no=2  4to orden
C             no=3  6to orden
C             no=4  8vo orden
C----------------------------------------------------------------------
*/
      int no=4;//Cambiarlo tambien en el kernel
      int ms=no+1;
/*
C----------------------------------------------------------------------
C     Par�metros generales
C     Unidades: Nx, Nz: [adimensional]
C               dh: [m]
C               dt: [seg]
C                v: [m/s]
C----------------------------------------------------------------------
*/
      int Nx=801;//1096,10000,5000
      int Nz=401;//401,2000
      float dh=25.0;
      int nt=4002;//4002,40
      float dt=0.002f;
/*C----------------------------------------------------------------------
C     Par�metros de frontera absorbente
C----------------------------------------------------------------------
C     N�mero de elementos absorbentes igual o mayor que el orden "no"*/
      int Na=25;//25,500

//C     Orden del polinomio de absorci�n
      float da=2;

//C     Par�metros que se cumplen para la transformaci�n S=ak+Q/(aa+iw):
//C     ak>1 y aa>0. Si ak=1 y aa=0 entonces se tiene la PML original
      //float ak=1.0;
      float aa=0.5;
/*
C----------------------------------------------------------------------
C     Par�metros de las fuentes (posicion y frecuencia media)
C----------------------------------------------------------------------
C     Unidades ixs, izs, ibs: [puntos de malla]
C                       nixs: [adimensional]
C                       fmid: [Hz]
C----------------------------------------------------------------------*/
      int ixs=1;//                               !Posici�n inicial horizontal
	  int izs=3;//                               !Profundidad
      int ibs=1;//                               !Incremento
      int nixs=Nx;//                             !N�mero de fuentes
      float fmid=25.0;//                           !Frecuencia media
	
/*	C----------------------------------------------------------------------
C     La posicion de los receptores es igual que la de las fuentes
C     s�lo ajusta su profundidad
C----------------------------------------------------------------------*/
	ixs=ixs+Na;
	Ajusta_profundidad(Na,no);
/*************************************************************************/
/*	C----------------------------------------------------------------------
C     Fuente tipo Ricker 2D de duraci�n limitada
C----------------------------------------------------------------------*/
	float *ricker;
	int nrick=(int)floor(1/(fmid*dt)+0.5);
	printf("nrick %d\n",nrick);
      //Crear memoria para el vector ricker
	  //ALLOCATE(ricker(nrick))
	ricker=asigna_memoria_vector_f(nrick+1);
	Fuente_Ricker(ricker,nrick,fmid,dt);
	/*for(int i=0;i<nrick;i++)
		printf("%.15E \n",ricker[i]);*/
/*************************************************************************/      
/*C----------------------------------------------------------------------
C     Dimensiones de la malla computacional
C----------------------------------------------------------------------*/
	int Nzz=Nz+Na;                                       //!No. puntos en Z
    int Nxx=Nx+2*Na;                                     //!No. puntos en X
//C----------------------------------------------------------------------
/*C----------------------------------------------------------------------
C     Se llena el vector de enteros de pasos
C----------------------------------------------------------------------*/
	int *mn,*ma;
	//ALLOCATE(mn(Nzz))
	mn=asigna_memoria_vector_int(Nzz+1);
	//ALLOCATE(ma(Na))
	ma=asigna_memoria_vector_int(Na+1);
	Llenar_vect_enteros_pasos(mn,ma,no,Nzz,Na);
	/*printf("\n");
	for(int i=0;i<Nzz;i++)
		printf("%d ",mn[i]);
	printf("\n");
	for(int i=0;i<Na;i++)
		printf("%d ",ma[i]);*/
//C----------------------------------------------------------------------
/*C----------------------------------------------------------------------
C     Se estima el vector de orden
C----------------------------------------------------------------------*/
    //ALLOCATE(g(ms))
	//ALLOCATE(d(no))
	float *g,*d;
	//float g[6],d[6];
    g=asigna_memoria_vector_f(ms+1);
	d=asigna_memoria_vector_f(no+1);
	
	Estimando_vect_orden(g,d,no);
    /*printf("\n");
	for(int i=0;i<ms;i++)
		printf("%.15E ",g[i]);
	printf("\n");
	for(int i=0;i<no;i++)
		printf("%.15E ",d[i]);*/
//C----------------------------------------------------------------------
/*C----------------------------------------------------------------------
C     Modelo de velocidades
C----------------------------------------------------------------------*/
      //ALLOCATE(vel(Nzz,Nxx))
	float *mvel,**vel;
	mvel=asigna_memoria_total_float(Nzz+1,Nxx+1);
	vel=asigna_memoria_matriz_float(mvel,Nzz+1,Nxx+1);
	/*Inicializar Vel*/

	int i,j;
	int CHUNK=20;//Nzz=Nz+Na=401+25
//Esto le da informacion adicional al compilador para que comparta los datos listados entre todos los hilos
#pragma omp parallel shared(Nzz,Nxx,vel) private(i,j)
	{
#pragma omp for schedule(dynamic,CHUNK) nowait
		for(i=0;i<=Nzz;i++){
			for(j=0;j<Nxx;j++){
				vel[i][j]=0.0;
			}
		}
	}
	/*for(int i=1;i<26;i++){
		for(int j=1;j<3;j++){
			printf("%.2E ",vel[j][i]);
		}
		printf("\n");
	}*/
	Modelo_velocidades(vel,Nx,Nz);
	/*for(int i=1;i<26;i++){
		for(int j=1;j<3;j++){
			printf("%.2E ",vel[j][i]);
		}
		printf("\n");
	}*/
	/*for(int i=0;i<10;i++){
		for(int j=0;j<10;j++){
			printf("%f ",vel[j][i]);
		}
		printf("\n");
	}*/
//C----------------------------------------------------------------------
/*C----------------------------------------------------------------------
C     Condici�n de estabilidad num�rica
C----------------------------------------------------------------------*/
	float tx,vmax;
	tx=Estabilidad_numerica(vel,Nx,Nz,dt,dh,&vmax);  
//C----------------------------------------------------------------------
/*C----------------------------------------------------------------------
C     Se hace vel=(dt^2/dx^2)*vel^2 y se expande la malla computacional
C----------------------------------------------------------------------*/
    /*for(int i=1;i<26;i++){
		for(int j=1;j<3;j++){
			printf("%.2E ",vel[j][i]);
		}
		printf("\n");
	}*/
	Modif_vel_Expandir_malla(vel,Nx,Nz,Na,Nxx,Nzz,tx);
	/*for(int i=1;i<26;i++){
		for(int j=1;j<3;j++){
			printf("%.2E ",vel[j][i]);
		}
		printf("\n");
	}*/
	
	//printf("%f",vel[Nzz-1][Nxx-1]);
//C----------------------------------------------------------------------
/*C----------------------------------------------------------------------
C     Despliegue en pantalla de los datos generales de la SOC generada
C----------------------------------------------------------------------*/
      printf("\n******************************************************\n");
      printf("    DATOS DE LA SECCION SISMICA EN TIEMPO GENERADA    \n");
      printf("             No. Trazas: %d\n",Nx);
      printf(" No. Muestras por traza: %d\n",nt-2);
      printf("   Intervalo de muestreo:  %f [seg]\n",dt);
      printf("\n******************************************************\n");
/*C      pause
C----------------------------------------------------------------------*/
      /*ALLOCATE(p(2,Nzz,Nxx))
      ALLOCATE(Ox(2,Nzz,Na))
      ALLOCATE(ux(2,Nzz,Na))
      ALLOCATE(Oz(Na,Nxx))
      ALLOCATE(uz(Na,Nxx))
      ALLOCATE(mr(no))
      ALLOCATE(ml(no))
      ALLOCATE(nr(no))
      ALLOCATE(nl(no))
      ALLOCATE(nb(no))
      ALLOCATE(Tn(Na))
      ALLOCATE(Tk(Na))
      ALLOCATE(Tt(Na))

	  INTEGER,ALLOCATABLE,DIMENSION(:) :: nl,nr,nb,mn,ma,ml,mr
		float PRECISION,ALLOCATABLE,DIMENSION(:) :: Tn,Tk,Tt,ricker,d,g
		float PRECISION,ALLOCATABLE,DIMENSION(:,:) :: vel,uz,Oz
		float PRECISION,ALLOCATABLE,DIMENSION(:,:,:) :: p,ux,Ox
	  */
	float *m1p,**m2p,***p;
	float *m1Ox,**m2Ox,***Ox;
	float *m1ux,**m2ux,***ux;

	/*float *mp1,**p1,*mp2,**p2;
	float *mOx1,**Ox1,*mOx2,**Ox2;
	float *mux1,**ux1,*mux2,**ux2;*/
	float *mOz,**Oz,*muz,**uz;
	int *mr,*ml,*nr,*nl,*nb;
	float *Tn,*Tk,*Tt;

	m1p=asigna_memoria_total_float(3,Nzz+1,Nxx+1);
	m2p=asigna_memoria_matriz_float(m1p,3,Nzz+1,Nxx+1);
	p=asigna_memoria_3D_float(m2p,3,Nzz+1,Nxx+1);
	/*mp1=asigna_memoria_total_float(Nzz+1,Nxx+1);
	p1=asigna_memoria_matriz_float(mp1,Nzz+1,Nxx+1);
	mp2=asigna_memoria_total_float(Nzz+1,Nxx+1);
	p2=asigna_memoria_matriz_float(mp2,Nzz+1,Nxx+1);*/

	m1Ox=asigna_memoria_total_float(3,Nzz+1,Na+1);
	m2Ox=asigna_memoria_matriz_float(m1Ox,3,Nzz+1,Na+1);
	Ox=asigna_memoria_3D_float(m2Ox,3,Nzz+1,Na+1);
	/*mOx1=asigna_memoria_total_float(Nzz+1,Na+1);
	Ox1=asigna_memoria_matriz_float(mOx1,Nzz+1,Na+1);
	mOx2=asigna_memoria_total_float(Nzz+1,Na+1);
	Ox2=asigna_memoria_matriz_float(mOx2,Nzz+1,Na+1);*/

	m1ux=asigna_memoria_total_float(3,Nzz+1,Na+1);
	m2ux=asigna_memoria_matriz_float(m1ux,3,Nzz+1,Na+1);
	ux=asigna_memoria_3D_float(m2ux,3,Nzz+1,Na+1);
	/*mux1=asigna_memoria_total_float(Nzz+1,Na+1);
	ux1=asigna_memoria_matriz_float(mux1,Nzz+1,Na+1);
	mux2=asigna_memoria_total_float(Nzz+1,Na+1);
	ux2=asigna_memoria_matriz_float(mux2,Nzz+1,Na+1);*/

	mOz=asigna_memoria_total_float(Na+1,Nxx+1);
	Oz=asigna_memoria_matriz_float(mOz,Na+1,Nxx+1);
	muz=asigna_memoria_total_float(Na+1,Nxx+1);
	uz=asigna_memoria_matriz_float(muz,Na+1,Nxx+1);
	
	mr=asigna_memoria_vector_int(no+1);
	ml=asigna_memoria_vector_int(no+1);
	nr=asigna_memoria_vector_int(no+1);
	nl=asigna_memoria_vector_int(no+1);
	nb=asigna_memoria_vector_int(no+1);
	
	Tn=asigna_memoria_vector_f(Na+1);
	Tk=asigna_memoria_vector_f(Na+1);
	Tt=asigna_memoria_vector_f(Na+1);


	
	Calculo_Ec_Onda(p,Ox,ux,
					uz,Oz,mvel,vel,
					Tn,Tk,Tt,ricker,g,d,
					mr,ml,nr,nl,nb,mn,ma,
					dt,dh,vmax,da,aa,tx,fmid,
					ixs,nixs,Na,Nxx,Nzz,no,izs,ms,nt,ibs,nrick);
/*	----------------TEMPORALMENTE BLOQUEADO MIENTRAS SE ARREGLA EL SYSTEMTIME
	GetSystemTime(&st_fin);
	printf("\nTime_ini %d : %d : %d : %d",st_ini.wHour,st_ini.wMinute,st_ini.wSecond,st_ini.wMilliseconds);
	printf("\nTime_fin %d : %d : %d : %d\n",st_fin.wHour,st_fin.wMinute,st_fin.wSecond,st_fin.wMilliseconds);
*/
	printf("Vamos Bien...\n");
	getchar();

	//Liberar Memoria
	free(ricker);free(mn);free(ma);free(g);free(d);free(mvel);free(vel);
	free(m1p);free(m2p);free(p);
	free(m1Ox);free(m2Ox);free(Ox);
	free(m1ux);free(m2ux);free(ux);
	free(mOz);free(Oz);free(muz);free(uz);free(mr);free(ml);
	free(nr);free(nl);free(nb);free(Tn);free(Tk);free(Tt);
	/*free(mp1);free(p1);free(mp2);free(p2);
	free(mOx1);free(Ox1);free(mOx2);free(Ox2);
	free(mux1);free(ux1);free(mux2);free(ux2);*/
	
}
void Ajusta_profundidad(int Na,int no){
	if (Na<no){ 
		 printf("Error. Se debe cumplir: Na>no\n");
		 getchar();
		 exit(0);
	}
	if ((no<1)||(4<no)){
		printf("Reajustar valor de no\n");
		getchar();
		exit(0);
	}
}
void Fuente_Ricker(float *ricker,int nrick,float fmid,float dt){
      float pi=4.0f * atan(1.0f);
      float cf=-2.0f * pi*pi*fmid*fmid;
	  float vmax,dl;
	  int i;
#pragma omp parallel shared(nrick,dt,cf,ricker) private(i,vmax,dl)
	{
#pragma omp for schedule(dynamic,2) nowait
		for(i=1;i<=nrick;i++){
		  vmax=(i-1)*dt;
		  dl=vmax*vmax;
		  ricker[i]=vmax*exp(cf*dl);
		}
	}

}
void Llenar_vect_enteros_pasos(int *mn,int *ma,int no,int Nzz,int Na){
	int i,j,k;
	for(i=1;i<=no;i++)
		mn[i]=i-1;
	
	for(j=i;j<=Nzz-Na;j++)
		mn[j]=no;

	k=Na;
	for(i=1;i<=no;i++){
		ma[k]=i-1;
		k=k-1;
	}
	for(j=i;j<=Na;j++){
		ma[k]=no;
		k=k-1;
	}
}
void Estimando_vect_orden(float *g,float *d,int no){
	switch (no){
		case 1:
			d[1]=1.0f/2.0f;
			g[1]=1.0f;
			g[2]=-2.0f;	
		break;
		case 2:
			d[1]=2.0f/3.0f;      d[2]=-1.0f/12.0f;
			g[1]=4.0f/3.0f;      g[2]=-1.0f/12.0f;
			g[3]=-5.0f/2.0f;
		break;
		case 3:
			d[1]=3.0f/4.0f;  d[2]=-3.0f/20.0f;  d[3]=1.0f/60.0f;
			g[1]=3.0f/2.0f;  g[2]=-3.0f/20.0f;  g[3]=1.0f/90.0f;
			g[4]=-49.0f/18.0f;
		break;
		case 4:
			d[1]=4.0f/5.0f; d[2]=-1.0f/5.0f; d[3]=4.0f/105.0f; d[4]=-1.0f/280.0f;
			g[1]=8.0f/5.0f; g[2]=-1.0f/5.0f; g[3]=8.0f/315.0f; g[4]=-1.0f/560.0f;
			g[5]=-205.0f/72.0f;
		break;
	
	
	}
}

void Modelo_velocidades(float **vel,int Nx,int Nz){
	 
	FILE *fp;
	int i,j;
	//open (UNIT=10,FILE='ModVelTabal.VEL',ACTION='Read',STATUS='old')
	if((fp=fopen("VelInt01_2D_IL_75.VEL","rb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	for(i=1;i<=Nx;i++){
		for(j=1;j<=Nz;j++){
			fscanf(fp,"%f",&vel[j][i]);
		}
	}
	//close(10)	
	fclose(fp);

	/*
	if((fp=fopen("ModVelTabal.VEL","r"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	FILE *fp2;
	if((fp2=fopen("ModVelTabal2.VEL","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	float val;
	while(!feof(fp)){
		fscanf(fp,"%f",&val);
		fprintf(fp2,"%f \n",val);
	} 

	fclose(fp);
	fclose(fp2);
	*/

}
float Estabilidad_numerica(float **vel,int Nx,int Nz,float dt,float dh,float *vmax){
	
	*vmax=-1.0;
	int i,j;
	for(i=1;i<=Nx;i++){
		for(j=1;j<=Nz;j++){
			if(vel[j][i]>*vmax)
				*vmax=vel[j][i];
		}
	}
    float tx=dt/dh;
	float cf=*vmax*sqrt(2.0f)*tx;
	//printf("%f  %E  %f",vmax,tx,cf);
	if (1.0f<cf){
          printf("Inestabilidad detectada 1<vmax*sqrt(2)*dt/dh= %f",cf);
          printf("Incrementa dh o disminuye dt");
          getchar();
		  exit(0);
	}
	return(tx);
}
void Modif_vel_Expandir_malla(float **vel,int Nx,int Nz,int Na,int Nxx,int Nzz,float tx){
	
	int i,j,k;

#pragma omp parallel shared(Nz,Nx,vel,tx) private(i,j)
	{
#pragma omp for schedule(dynamic,20) nowait
		for(i=1;i<=Nz;i++){
			for(j=1;j<=Nx;j++){
				vel[i][j]=(vel[i][j]*vel[i][j])*(tx*tx);
			}
		}
	}
	int H=Nx+Na;
	for(i=H;i>Na;i--){
		for(j=1;j<=Nz;j++){
			vel[j][i]=vel[j][i-Na];
		
		}
		for(k=j;k<=Nzz;k++){
			vel[k][i]=vel[Nz][i-Na];
		}
	}
#pragma omp parallel shared(Na,Nz,Nzz,vel) private(i,j)
	{
#pragma omp for schedule(dynamic,2) nowait
		for(i=1;i<=Na;i++){
			for(j=Nz+1;j<=Nzz;j++){
				vel[j][i]=vel[Nz][i];
			}
		}
	}
	for(i=Nxx;i>H;i--){
		for(j=1;j<=Nz;j++){
			vel[j][i]=vel[j][i-Na];
		}
		for(k=j;k<=Nzz;k++){
			vel[k][i]=vel[Nz][i-Na];
		}
	}
}


void Calculo_Ec_Onda(float ***p,float ***Ox,float ***ux,
					 float **uz,float **Oz,float *mvel,float **vel,
					 float *Tn,float *Tk,float *Tt,float *ricker,float *g,float *d,
					 int *mr,int *ml,int *nr,int *nl,int *nb,int *mn,int *ma,
					 float dt,float dh,float vmax,float da,float aa,float tx,float fmid,
					 int ixs,int nixs,int Na,int Nxx,int Nzz,int no,int izs,int ms,int nt,int ibs,int nrick){
/*C----------------------------------------------------------------------
C     Repetici�n del c�lculo de la soluci�n de la ecuaci�n de onda
C      (se resuelve la ecuaci�n de onda para cada fuente-receptor)
C----------------------------------------------------------------------*/

		float *p_host,*p_host2,*Ox_host,*ux_host,*Oz_host,*uz_host;
		p_host=asigna_memoria_vector_f(2*(Nzz+1)*(Nxx+1));
		p_host2=asigna_memoria_vector_f(Nxx-2*Na);
		Ox_host=asigna_memoria_vector_f(2*(Nzz+1)*(Na+1));
		ux_host=asigna_memoria_vector_f(2*(Nzz+1)*(Na+1));
		Oz_host=asigna_memoria_vector_f((Na+1)*(Nxx+1));
		uz_host=asigna_memoria_vector_f((Na+1)*(Nxx+1));

	




	float tx2=dt/(dh*dh);
    //open(12,file='STFM_Tabal_PML.RES')
	FILE *fp,*fp2;
	if((fp=fopen("../../STFM_Yax_PML_GPU_OpenMP_float.RES","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	
    //int ixo=ixs;
	//int nF=1;
	float dl,pi,cf;
	int i,j,k1,k2,N,M,L;
	//int h,k;
	int i_time;

	//for(nF=1;nF<=nixs;nF++){
		//printf("Grabando fuente-receptor: %d \n",nF);
		/*	!----------------------------------------------------------------
		  !  Coeficientes de fronteras absorbentes
		  !  Qi(l)=Qmax*(l/d)^m
		  !  Qmax=(3*vp*log(1/R))/(2*d)
		  !  vp: velocidad, d: longitud de la PML, R: reflexi�n te�rica
		  !----------------------------------------------------------------	*/
		dl=Na*dh;
		pi=3.0f*vmax/(2.0f*dl);
		cf=pi*log(10000.0f);
		pi=dl;
		for(i=1;i<=Na;i++){  
			Tn[i]=pow((pi/dl),da);//P
			Tn[i]=cf*Tn[i];//P
			pi=pi-dh;//Acum pi=pi-Na*dh
		}
		
		//Creamos memoria en device
		float *Tn_dev,*Tk_dev,*Tt_dev;
//--- cuda safe call ya no es viable
/*		CUDA_SAFE_CALL(hipMalloc((void **)&Tn_dev,(Na+1)*sizeof(float)));
		CUDA_SAFE_CALL(hipMalloc((void **)&Tk_dev,(Na+1)*sizeof(float)));
		CUDA_SAFE_CALL(hipMalloc((void **)&Tt_dev,(Na+1)*sizeof(float)));
*/
		checkCudaErrors(hipMalloc((void **)&Tn_dev,(Na+1)*sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&Tk_dev,(Na+1)*sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&Tt_dev,(Na+1)*sizeof(float)));

		//CUDA_SAFE_CALL(hipMemcpy(Tn_dev,Tn,(Na+1)*sizeof(float),hipMemcpyHostToDevice));
		dim3 blockNa(16);
		dim3 gridNa(iDivUp(Na+1,blockNa.x));
		/*Calcular_Tn<<<gridNa,blockNa>>>(Tn_dev,pi,dl,da,cf,Na+1);
		pi=pi-Na*dh;*/


		//CUDA_SAFE_CALL(hipMemcpy(Tn,Tn_dev,(Na+1)*sizeof(float),hipMemcpyDeviceToHost));
//---CUIDADO CON EL
		checkCudaErrors(hipMemcpy(Tn_dev,Tn,(Na+1)*sizeof(float),hipMemcpyHostToDevice));


		//printf("%f %f %f\n",dl,pi,cf);
		/*!----------------------------------------------------------------
      !  C�lculo de: Tk = Tn + aa  y: Tt=dt*(Tk+1)
      !----------------------------------------------------------------*/
        dl=dh*dh;
		for(i=1;i<=Na;i++){	
			cf=Tn[i] + aa;//No acum...
            Tk[i]=cf*dl;//P
            Tt[i]=(1.0f - dt*cf);//P
		}
		//Calcular_Tk_Tt<<<gridNa,blockNa>>>(Tk_dev,Tt_dev,Tn_dev,aa,dl,dt,Na+1);
		//CUDA_SAFE_CALL(hipMemcpy(Tk,Tk_dev,(Na+1)*sizeof(float),hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(Tt,Tt_dev,(Na+1)*sizeof(float),hipMemcpyDeviceToHost));

		checkCudaErrors(hipMemcpy(Tk_dev,Tk,(Na+1)*sizeof(float),hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(Tt_dev,Tt,(Na+1)*sizeof(float),hipMemcpyHostToDevice));

		k1=1;
        k2=2;
	
		float *p_dev,*p_dev2,*Ox_dev,*ux_dev;
		size_t size1= 2*(Nzz+1)*(Nxx+1)*sizeof(float);
		checkCudaErrors(hipMalloc((void **)&p_dev,size1));
		checkCudaErrors(hipMalloc((void **)&p_dev2,(Nxx-2*Na)*sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&Ox_dev,2*(Nzz+1)*(Na+1)*sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&ux_dev,2*(Nzz+1)*(Na+1)*sizeof(float)));
		
		dim3 blockNzzNxx(16,16,2);
		dim3 gridNzzNxx(iDivUp(Nzz+1,blockNzzNxx.x),iDivUp(Nxx+1,blockNzzNxx.y),1);
		Inicializar_p<<<gridNzzNxx,blockNzzNxx>>>(p_dev,(Nzz+1),(Nxx+1),2);
		dim3 blockNzzNa(16,16,2);
		dim3 gridNzzNa(iDivUp(Nzz+1,blockNzzNa.x),iDivUp(Na+1,blockNzzNa.y),1);
		Inicializar_Ox_ux<<<gridNzzNa,blockNzzNa>>>(Ox_dev,ux_dev,(Nzz+1),(Na+1),2);


		hipChannelFormatDesc description = hipCreateChannelDesc<float>();
		checkCudaErrors(hipMallocArray(&cu_array_p1, &description, Nxx+1,Nzz+1));
		checkCudaErrors(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
		checkCudaErrors(hipBindTextureToArray(tex_p1,cu_array_p1));
		
		checkCudaErrors(hipMallocArray(&cu_array_p2, &description, Nxx+1,Nzz+1));
		checkCudaErrors(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
		checkCudaErrors(hipBindTextureToArray(tex_p2,cu_array_p2));
		


		float *uz_dev,*Oz_dev;
		checkCudaErrors(hipMalloc((void **)&uz_dev,(Na+1)*(Nxx+1)*sizeof(float)));
		checkCudaErrors(hipMalloc((void **)&Oz_dev,(Na+1)*(Nxx+1)*sizeof(float)));
		
		dim3 blockNaNxx(16,16);
		dim3 gridNaNxx(iDivUp(Na+1,blockNaNxx.x),iDivUp(Nxx+1,blockNaNxx.y));
		Inicializar_uz_Oz<<<gridNaNxx,blockNaNxx>>>(uz_dev,Oz_dev,(Na+1),(Nxx+1));

		/*
		CUDA_SAFE_CALL(hipMemcpy(p_host,p_dev,2*(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(Ox_host,Ox_dev,2*(Nzz+1)*(Na+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(ux_host,ux_dev,2*(Nzz+1)*(Na+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(Oz_host,Oz_dev,(Na+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(uz_host,uz_dev,(Na+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		
		Pasar_Inf_Var(p,Ox,ux,Oz,uz,p_host,Ox_host,ux_host,Oz_host,uz_host,Nzz,Nxx,Na);
		Guardar_Info_Var(p,Ox,ux,Oz,uz,Nzz,Nxx,Na);
		*/
		
		float *vel_dev;
		checkCudaErrors(hipMalloc((void **)&vel_dev,(Nzz+1)*(Nxx+1)*sizeof(float)));
		checkCudaErrors(hipMemcpy(vel_dev,mvel,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyHostToDevice));

		//hipChannelFormatDesc description = hipCreateChannelDesc<float>();
		checkCudaErrors(hipMallocArray(&cu_array_vel, &description, Nxx+1,Nzz+1));
		checkCudaErrors(hipMemcpyToArray(cu_array_vel,0,0,mvel,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyHostToDevice));
		checkCudaErrors(hipBindTextureToArray(tex_vel,cu_array_vel));
		
		/*CUDA_SAFE_CALL(hipMemcpy(mvel,vel_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		
		if((fp2=fopen("../../vel_parallel1.txt","wb"))==NULL){
			printf("\n No se puede abrir el archivo...");
			getchar();
			exit(0);
		}

		for(int i=0;i<=Nzz;i++){
			for(int j=0;j<Nxx;j++){
				fprintf(fp2,"%lf \n",mvel[i*Nxx+j]);
			}
		}
		fclose(fp2);

		Pasar_Valores_vel<<<gridNzzNxx,blockNzzNxx>>>(vel_dev,Nzz+1,Nxx+1);
		CUDA_SAFE_CALL(hipMemcpy(mvel,vel_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));

		if((fp2=fopen("../../vel_parallel2.txt","wb"))==NULL){
			printf("\n No se puede abrir el archivo...");
			getchar();
			exit(0);
		}

		for(int i=0;i<=Nzz;i++){
			for(int j=0;j<Nxx;j++){
				fprintf(fp2,"%lf \n",mvel[i*Nxx+j]);
			}
		}
		fclose(fp2);*/

		/*for(i=1;i<=no;i++){
			mr[i]=0;
            ml[i]=0;
			nr[i]=0;
            nl[i]=0;
			nb[i]=0;
		}*/
		i_time=1;
        /*//!  Fuente �nica
        p[k1][izs][ixo]=ricker[i_time];*/
		/*C----------------------------------------------------------------------
		C     Fuentes m�ltiples
		C----------------------------------------------------------------------*/
		/*ixo=ixs;
		for(i=1;i<=nixs;i++){//PD
			p[k1][izs][ixo]=ricker[i_time];
			ixo=ixo+ibs;//Posici�n
		}*/
		float *ricker_dev;
		checkCudaErrors(hipMalloc((void **)&ricker_dev,(Na+1)*sizeof(float)));
		checkCudaErrors(hipMemcpy(ricker_dev,ricker,(Na+1)*sizeof(float),hipMemcpyHostToDevice));

		/*for(i=1;i<=Na;i++)
			printf("%f \n",ricker[i]);*/
		
		/*dim3 block_nixs(nixs+1);
		dim3 grid_1(1);*/
		dim3 block_nixs(16);
		dim3 grid_1(iDivUp(nixs+1,block_nixs.x));
		
		Fuentes_multiples<<<grid_1,block_nixs>>>(p_dev,ricker[i_time],ricker[i_time+1],izs,ixs,ibs,nixs+1,Nzz+1,Nxx+1);
		
		//C----------------------------------------------------------------------
        i_time=2;
		/*//!  Fuente �nica
        p[k2][izs][ixo]=ricker[i_time];*/
		/*C----------------------------------------------------------------------
		C     Fuentes m�ltiples
		C----------------------------------------------------------------------*/
		/*ixo=ixs;
		for(i=1;i<=nixs;i++){//PD
			p[k2][izs][ixo]=ricker[i_time];
			ixo=ixo+ibs;//Posici�n
		}*/
		//C----------------------------------------------------------------------

		int *mn_dev,*ma_dev;
		checkCudaErrors(hipMalloc((void **)&mn_dev,(Nzz+1)*sizeof(int)));
		checkCudaErrors(hipMalloc((void **)&ma_dev,(Na+1)*sizeof(int)));
		checkCudaErrors(hipMemcpy(mn_dev,mn,(Nzz+1)*sizeof(int),hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(ma_dev,ma,(Na+1)*sizeof(int),hipMemcpyHostToDevice));

		hipChannelFormatDesc description2 = hipCreateChannelDesc<int>();
		checkCudaErrors(hipMallocArray(&cu_array_mn, &description2,Nzz+1,1));
		checkCudaErrors(hipMemcpyToArray(cu_array_mn,0,0,mn,(Nzz+1)*sizeof(int),hipMemcpyHostToDevice));
			
		checkCudaErrors(hipMallocArray(&cu_array_ma, &description2,Na+1,1));
		checkCudaErrors(hipMemcpyToArray(cu_array_ma,0,0,ma,(Na+1)*sizeof(int),hipMemcpyHostToDevice));
		
		/*printf("mn 1: \n ");
		for(int i=0;i<Nzz+1;i++){
			printf("%d ",mn[i]);
		}*/

		//CUDA_SAFE_CALL(hipBindTexture(0,tex_mn,mn_dev,(Nzz+1)*sizeof(int)));
		//CUDA_SAFE_CALL(hipBindTexture(0,tex_ma,ma_dev,(Na+1)*sizeof(int)));
		checkCudaErrors(hipBindTextureToArray(tex_mn,cu_array_mn));
		checkCudaErrors(hipBindTextureToArray(tex_ma,cu_array_ma));

		/*dim3 sizeblockmn(20);
		dim3 gridmn(iDivUp(Nzz+1,sizeblockmn.x));
		Pasar_Valores_vel<<<gridmn,sizeblockmn>>>(mn_dev,Nzz+1);
		CUDA_SAFE_CALL(hipMemcpy(mn,mn_dev,(Nzz+1)*sizeof(int),hipMemcpyDeviceToHost));
		*/
		/*printf("mn 2: \n ");
		for(int i=0;i<Nzz+1;i++){
			printf("%d ",mn[i]);
		}*/
		

		/*float *g_dev,*d_dev;
		CUDA_SAFE_CALL(hipMalloc((void **)&g_dev,(ms+1)*sizeof(float)));
		CUDA_SAFE_CALL(hipMalloc((void **)&d_dev,(no+1)*sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpy(g_dev,g,(ms+1)*sizeof(float),hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_dev,d,(no+1)*sizeof(float),hipMemcpyHostToDevice));*/
		
		/*for(int i=1;i<=ms;i++){
			printf("%f ",g[i]);
		}
		printf("\n");*/
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_dev),g, (ms+1)*sizeof(float),0,hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_dev),d, (no+1)*sizeof(float),0,hipMemcpyHostToDevice));
		/*hipMemcpyFromSymbol(g,HIP_SYMBOL(g_dev),6*sizeof(float),0,hipMemcpyDeviceToHost); 
		for(int i=1;i<=ms;i++){
			printf("%f ",g[i]);
		}*/

		//!  Se calculan las funciones absorbentes
		//!  Para ux y uz
        N=Nxx;
        M=Nzz;
		
		
		dim3 blockNzz(16);
		dim3 gridNzz(iDivUp(Nzz+1,blockNzz.x));
		dim3 blockNxx(16);
		dim3 gridNxx(iDivUp(Nxx+1,blockNxx.x));
		/*
		Nzz-Na
		M=Na+1;
		N=Nxx-Na;
		*/
		dim3 blockNzzNaMN(16,16);
		dim3 gridNzzNaMN(iDivUp(Nzz-Na+1,blockNzzNaMN.x), iDivUp(Nxx-2*Na,blockNzzNaMN.y));

		dim3 blockNaNa(16,16);
		dim3 gridNaNa(iDivUp(Na+1,blockNaNa.x), iDivUp(Na+1,blockNaNa.y));

		dim3 blockNaNxxNa(16,16);
		dim3 gridNaNxxNa(iDivUp(Na+1,blockNaNxxNa.x), iDivUp(Nxx-2*Na,blockNaNxxNa.y));
		

		dim3 blockNzzNaNa(16,16);
		dim3 gridNzzNaNa(iDivUp(Nzz-Na+1,blockNzzNaNa.x), iDivUp(Na+1,blockNzzNaNa.y));

		//SYSTEMTIME st_ini,st_fin;

		//GetSystemTime(&st_ini);
		
		dim3 blockNaNzz(16,16);
		dim3 gridNaNzz(iDivUp(Na+1,blockNaNzz.x),iDivUp(Nzz+1,blockNaNzz.y));

		Calcular_funciones_absorbentes1<<<gridNaNzz,blockNaNzz>>>(p_dev,Ox_dev,ux_dev,Tn_dev,Tk_dev,
																 //mn_dev,ma_dev,//g_dev,d_dev,
																 tx2,dh,
																 N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1);
		
		/*dim3 blockNaNxx(16,16);
		dim3 gridNaNxx(iDivUp(Na+1,blockNaNzz.x),iDivUp(Nxx+1,blockNaNzz.y));*/
		Calcular_funciones_absorbentes2<<<gridNaNxx,blockNaNxx>>>(p_dev,Oz_dev,uz_dev,Tn_dev,Tk_dev,
																 //mn_dev,ma_dev,//g_dev,d_dev,
																 tx2,dh,
																 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1);
		
		
		
		/*GetSystemTime(&st_fin);
		printf("\nTime_ini %d : %d",st_ini.wSecond,st_ini.wMilliseconds);
		printf("\nTime_fin %d : %d\n",st_fin.wSecond,st_fin.wMilliseconds);

		getchar();*/

		i_time=3;

		dim3 blockNxxNa(16);
		dim3 gridNxxNa(iDivUp(Nxx-2*Na,blockNxxNa.x));

		/*SYSTEMTIME st_ini,st_fin;
		GetSystemTime(&st_ini);*/
		dim3 gridNaNxxNzz(iDivUp(Na+1,blockNaNzz.x),iDivUp(Nzz+1,blockNaNzz.y));
		if(Nxx>Nzz){
			//gridNaNxxNzz.x=iDivUp(Na+1,blockNaNzz.x);
			gridNaNxxNzz.y=iDivUp(Nxx+1,blockNaNzz.y);
		}
		dim3 gridNuevo1(iDivUp(Na+1,blockNaNzz.x),iDivUp(Nzz+1,blockNaNzz.y));
		if(Nxx-2*Na > Nzz+1)
			gridNuevo1.y=iDivUp(Nxx-2*Na,blockNaNzz.y);

		/*CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
		CUDA_SAFE_CALL(hipBindTextureToArray(tex_p1,cu_array_p1));				
		CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
		CUDA_SAFE_CALL(hipBindTextureToArray(tex_p2,cu_array_p2));*/
		//for(L=3;L<=3;L++){
		for(L=3;L<=nt;L++){
			//Para Ox y Oz
            N=Nxx;
            M=Nzz;
			//GetSystemTime(&st_ini);

			/*ParaOx<<<gridNaNzz,blockNaNzz>>>(p_dev,Ox_dev,Tt_dev,
											mn_dev,ma_dev,//g_dev,d_dev,
											tx,dh,
											N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1);
			
			ParaOz<<<gridNaNxx,blockNaNxx>>>(p_dev,Oz_dev,Tt_dev,
											 mn_dev,ma_dev,//g_dev,d_dev,
											 tx,dh,
											 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1);
			*/
			/*ParaOxOz<<<gridNaNxxNzz,blockNaNzz>>>(p_dev,Ox_dev,Oz_dev,Tt_dev,
											      mn_dev,ma_dev,
											      tx,dh,
											      N,M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1);
			*/
			
			ParaOxOz2<<<gridNaNxxNzz,blockNaNzz>>>(Ox_dev,Oz_dev,Tt_dev,
											      //mn_dev,ma_dev,
											      tx,dh,
											      N,M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1);
			
			/*GetSystemTime(&st_fin);
			printf("\nTime_ini %d : %d",st_ini.wSecond,st_ini.wMilliseconds);
			printf("\nTime_fin %d : %d\n",st_fin.wSecond,st_fin.wMilliseconds);
			getchar();*/

			//!     Se resuelve la ecuaci�n solo en el dominio real
			//!     Para u en n+2
			M=Na+1;
			N=Nxx-Na;
			
			/*CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
			CUDA_SAFE_CALL(hipBindTextureToArray(tex_p1,cu_array_p1));
			CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
			CUDA_SAFE_CALL(hipBindTextureToArray(tex_p2,cu_array_p2));*/

			/*EcDominioReal<<<gridNzzNaMN,blockNzzNaMN>>>(p_dev,Oz_dev,Tt_dev,//vel_dev,
											 mn_dev,ma_dev,//g_dev,d_dev,
											 tx,dh,
											 M,N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);
			*/
			
			EcDominioReal2<<<gridNzzNaMN,blockNzzNaMN>>>(p_dev,Oz_dev,Tt_dev,//vel_dev,
											 //mn_dev,ma_dev,//g_dev,d_dev,
											 tx,dh,
											 M,N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);
			
			
			//!     Se resuelve para el dominio ficticio
			//!     Se calculan las funciones absorbentes ux-uz
			//!     para la siguiente iteracion
            N=Nxx;
			//!     para ux y p en i_time=n+2

			
			/*if(k1==1){
				CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
				CUDA_SAFE_CALL(hipBindTextureToArray(tex_p1,cu_array_p1));
			}
			else{
				CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
				CUDA_SAFE_CALL(hipBindTextureToArray(tex_p2,cu_array_p2));
			}*/
			
			Para_uxp<<<gridNaNzz,blockNaNzz>>>(p_dev,Ox_dev,ux_dev,Tn_dev,Tk_dev,vel_dev,
											   //mn_dev,ma_dev,//g_dev,d_dev,
											   tx2,dh,
											   N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);
			/*Para_uxp2<<<gridNaNzz,blockNaNzz>>>(p_dev,Ox_dev,ux_dev,Tn_dev,Tk_dev,//vel_dev,
											   //mn_dev,ma_dev,//g_dev,d_dev,
											   tx2,dh,
											   N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);*/
			
			/*
			CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
			CUDA_SAFE_CALL(hipBindTextureToArray(tex_p1,cu_array_p1));
			CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
			CUDA_SAFE_CALL(hipBindTextureToArray(tex_p2,cu_array_p2));
			*/
			//!     para uz and p in itime=n+2
            M=Nzz;

			Para_uzp<<<gridNaNa,blockNaNa>>>(p_dev,Oz_dev,uz_dev,Tn_dev,Tk_dev,vel_dev,
											 //mn_dev,ma_dev,//g_dev,d_dev,
											 tx2,dh,
											 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);
			
			/*Para_uzp2<<<gridNaNa,blockNaNa>>>(p_dev,Oz_dev,uz_dev,Tn_dev,Tk_dev,//vel_dev,
											 //mn_dev,ma_dev,//g_dev,d_dev,
											 tx2,dh,
											 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);*/
			
			/*Para_uxp_uzp<<<gridNaNzz,blockNaNzz>>>(p_dev,Ox_dev,Oz_dev,ux_dev,uz_dev,Tn_dev,Tk_dev,vel_dev,
						                     mn_dev,ma_dev,
						                     tx2,dh,
						                     N,M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);
			*/
			/*Para_uxp_uzp<<<gridNuevo1,blockNaNzz>>>(p_dev,Ox_dev,Oz_dev,ux_dev,uz_dev,Tn_dev,Tk_dev,vel_dev,
						                     mn_dev,ma_dev,
						                     tx2,dh,
						                     N,M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1,Nxx-2*Na);
			*/
			
			/*Enmedio_uz<<<gridNaNxxNa,blockNaNxxNa>>>(p_dev,Oz_dev,uz_dev,Tn_dev,Tk_dev,vel_dev,
											 //mn_dev,ma_dev,//g_dev,d_dev,
											 tx2,dh,
											 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1,Nxx-2*Na);

			*/
			if(k1==1){
				checkCudaErrors(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
				checkCudaErrors(hipBindTextureToArray(tex_p1,cu_array_p1));
			}
			else{
				checkCudaErrors(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
				checkCudaErrors(hipBindTextureToArray(tex_p2,cu_array_p2));
			}			
			Enmedio_uz2<<<gridNaNxxNa,blockNaNxxNa>>>(p_dev,Oz_dev,uz_dev,Tn_dev,Tk_dev,
											 //mn_dev,ma_dev,//g_dev,d_dev,
											 tx2,dh,
											 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1,Nxx-2*Na);


			Final_uzp<<<gridNaNa,blockNaNa>>>(p_dev,Oz_dev,uz_dev,Tn_dev,Tk_dev,vel_dev,
											 //mn_dev,ma_dev,//g_dev,d_dev,
											 tx2,dh,
											 M,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);

			
			
			ParteSuperior_ux<<<gridNzzNaNa,blockNzzNaNa>>>(p_dev,Ox_dev,ux_dev,Tn_dev,Tk_dev,vel_dev,
											  // mn_dev,ma_dev,//g_dev,d_dev,
											   tx2,dh,
											   N,no,Na+1,Nzz+1,Nxx+1,ms,k1-1,k2-1);

			
			/*//!     Fuente �nica
			if(i_time<(1/(fmid*dt))){
				p[k1][izs][ixo]=ricker[i_time];
			}*/
			/*C----------------------------------------------------------------------
			C     Fuentes m�ltiples
			C----------------------------------------------------------------------*/
			/*if(i_time<nrick){
				//ixo=ixs;
				Fuentes_multiples2<<<grid_1,block_nixs>>>(p_dev,ricker_dev,izs,i_time,ixs,ibs,nixs+1,Nzz+1,Nxx+1,k1-1);
			}*/

			if(i_time<nrick){
				//ixo=ixs;
				Fuentes_multiples3<<<grid_1,block_nixs>>>(p_dev,ricker[i_time],izs,ixs,ibs,nixs+1,Nzz+1,Nxx+1,k1-1);
			}

			if(k1==1){
				checkCudaErrors(hipMemcpyToArray(cu_array_p1,0,0,p_dev,(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
				checkCudaErrors(hipBindTextureToArray(tex_p1,cu_array_p1));
			}
			else{
				checkCudaErrors(hipMemcpyToArray(cu_array_p2,0,0,p_dev+(Nzz+1)*(Nxx+1),(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToDevice));
				checkCudaErrors(hipBindTextureToArray(tex_p2,cu_array_p2));
			}


			/* !     Se imprime la secci�n en tiempo sin los puntos de frontera
			!     laterales y considerando �nicamente receptores en la
			!     superficie*/
            /*do J=Na+1,Nxx-Na
		         WRITE(12,*)p(k1,1,J)
			end do*/

			Pasar_Valores<<<gridNxxNa,blockNxxNa>>>(p_dev2,p_dev,Nxx-2*Na,Nxx+1,Na+1,Nzz+1,k1-1);
			//CUDA_SAFE_CALL(hipMemcpy(p_host,p_dev,2*(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(p_host2,p_dev2,(Nxx-2*Na)*sizeof(float),hipMemcpyDeviceToHost));
			
			/*for(i=1;i<=Nzz;i++){
				for(j=1;j<=Nxx;j++){
					p[1][i][j]=p_host[j*Nzz+i];
					p[2][i][j]=p_host[Nzz*Nxx+j*Nzz+i];
				}
			}*/

			/*for(j=Na+1;j<=Nxx-Na;j++){
				fprintf(fp,"%f \n",p_host[(k1-1)*(Nzz+1)*(Nxx+1) + (Nxx+1) + j]);
				//printf("%.15f \n",p_host[(k1-1)*(Nzz+1)*(Nxx+1) + (Nxx+1) + j]);
			}*/
			for(j=0;j<(Nxx-2*Na);j++)
				fprintf(fp,"%f \n",p_host2[j]);
			


			/*
			for(j=Na+1;j<=Nxx-Na;j++){
				fprintf(fp,"%f \n",p[k1][1][j]);
			}*/

			//printf("%d  %.15E \n",i_time,p[k1][1][ixo]);//WRITE(12,*)p(k1,1,ixo)
			//fprintf(fp,"%.15E \n",p[k1][1][ixo]);
            N=k1;
            k1=k2;
            k2=N;
			//WRITE(*,*)'Grabando tiempo',i_time
			printf("\nGrabando tiempo: %d",i_time);
            i_time=i_time+1;  

			

		}
		//ixo=ixo+ibs;
	//}	
		
	/*
		CUDA_SAFE_CALL(hipMemcpy(p_host,p_dev,2*(Nzz+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(Ox_host,Ox_dev,2*(Nzz+1)*(Na+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(ux_host,ux_dev,2*(Nzz+1)*(Na+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(Oz_host,Oz_dev,(Na+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(uz_host,uz_dev,(Na+1)*(Nxx+1)*sizeof(float),hipMemcpyDeviceToHost));
		
		Pasar_Inf_Var(p,Ox,ux,Oz,uz,p_host,Ox_host,ux_host,Oz_host,uz_host,Nzz,Nxx,Na);
		Guardar_Info_Var(p,Ox,ux,Oz,uz,Nzz,Nxx,Na);
	*/	
	/*GetSystemTime(&st_fin);
	printf("\nTime_ini %d : %d : %d : %d",st_ini.wHour,st_ini.wMinute,st_ini.wSecond,st_ini.wMilliseconds);
	printf("\nTime_fin %d : %d : %d : %d\n",st_fin.wHour,st_fin.wMinute,st_fin.wSecond,st_fin.wMilliseconds);
	*/

	fclose(fp);

	/*Liberar memoria del device*/
	hipFree(Tn_dev);
	hipFree(Tk_dev);
	hipFree(Tt_dev);

	hipFree(p_dev);hipFree(p_dev2);hipFree(Ox_dev);hipFree(ux_dev);
	hipFree(uz_dev);hipFree(Oz_dev);
	
	hipFree(vel_dev);

	hipFree(ricker_dev);
	hipFree(mn_dev);hipFree(ma_dev);
	/*hipFree(mr_dev);hipFree(ml_dev);
	hipFree(nr_dev);hipFree(nl_dev);hipFree(nb_dev);*/
	hipFreeArray(cu_array_vel);
	hipFreeArray(cu_array_p1);
	hipFreeArray(cu_array_p2);
	hipFreeArray(cu_array_mn);
	hipFreeArray(cu_array_ma);

	free(p_host);
	free(p_host2);
	free(Ox_host);free(ux_host);
	free(Oz_host);free(uz_host);

}

void Guardar_Info_Var(float ***p,float ***Ox,float ***ux,float **Oz,float **uz,int Nzz,int Nxx,int Na){
	FILE *fp_p,*fp_Ox,*fp_ux,*fp_Oz,*fp_uz;
	
	if((fp_p=fopen("../../p2.txt","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	if((fp_Ox=fopen("../../Ox2.txt","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	if((fp_ux=fopen("../../ux2.txt","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	if((fp_Oz=fopen("../../Oz2.txt","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	if((fp_uz=fopen("../../uz2.txt","wb"))==NULL){
		printf("\n No se puede abrir el archivo...");
		getchar();
		exit(0);
	}
	int i,j,k;

	for(k=1;k<=2;k++){
		for(i=1;i<=Nzz;i++){
			for(j=1;j<=Nxx;j++){
				fprintf(fp_p,"%f ",p[k][i][j]);

			}
			fprintf(fp_p,"\n");

			for(j=1;j<=Na;j++){
				fprintf(fp_Ox,"%f ",Ox[k][i][j]);
				fprintf(fp_ux,"%f ",ux[k][i][j]);
			}
			fprintf(fp_Ox,"\n");
			fprintf(fp_ux,"\n");

		}
		
		fprintf(fp_p,"\n\n");
		fprintf(fp_p,"Mitad\n\n");
		
		fprintf(fp_Ox,"\n\n");
		fprintf(fp_Ox,"Mitad\n\n");
		fprintf(fp_ux,"\n\n");
		fprintf(fp_ux,"Mitad\n\n");
	}

	for(i=1;i<=Na;i++){
		for(j=1;j<=Nxx;j++){
			fprintf(fp_Oz,"%f ",Oz[i][j]);
			fprintf(fp_uz,"%f ",uz[i][j]);
		}
		fprintf(fp_Oz,"\n");
		fprintf(fp_uz,"\n");
	}

	fclose(fp_p);
	fclose(fp_Ox);
	fclose(fp_ux);
	fclose(fp_Oz);
	fclose(fp_uz);
}

void Pasar_Inf_Var(float ***p,float ***Ox,float ***ux,float **Oz,float **uz,
				   float *p_host,float *Ox_host,float *ux_host,float *Oz_host,float *uz_host,
				   int Nzz,int Nxx,int Na){

	int i,j,k;

	for(k=1;k<=2;k++){
		for(i=1;i<=Nzz;i++){
			for(j=1;j<=Nxx;j++){
				p[k][i][j]=p_host[(k-1)*(Nzz+1)*(Nxx+1) + i*(Nxx+1) + j];
			}
			for(j=1;j<=Na;j++){
				Ox[k][i][j]=Ox_host[(k-1)*(Nzz+1)*(Na+1) + i*(Na+1) + j];
				ux[k][i][j]=ux_host[(k-1)*(Nzz+1)*(Na+1) + i*(Na+1) + j];
			}
		}	
	}
	for(i=1;i<=Na;i++){
		for(j=1;j<=Nxx;j++){
			Oz[i][j]=Oz_host[i*(Nxx+1)+j];
			uz[i][j]=uz_host[i*(Nxx+1)+j];	
		}
	}
}

 
